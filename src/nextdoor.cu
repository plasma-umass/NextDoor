#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <string>
#include <stdio.h>
#include <vector>
#include <bitset>
#include <unordered_set>
#include <time.h>
#include <sys/time.h>
#include <unistd.h>
#include <algorithm>
#include <numeric>
#include <string.h>
#include <assert.h>
#include <tuple>
#include <queue>
#include <cub/block/block_load.cuh>
#include <cub/block/block_store.cuh>
#include <cub/block/block_radix_sort.cuh>
#include <cub/block/block_scan.cuh>
#include <cub/device/device_select.cuh>
#include <hipcub/hipcub.hpp>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>

#include "sample.hpp"

#ifndef __NEXTDOOR_CU__
#define __NEXTDOOR_CU__

typedef VertexID VertexID_t;

#include "csr.hpp"
#include "utils.hpp"
#include "sampler.cuh"
#include "rand_num_gen.cuh"
#include "libNextDoor.hpp"

using namespace utils;
using namespace GPUUtils;

#define CHECK_RESULT

const size_t N_THREADS = 256;


const int ALL_NEIGHBORS = -1;

const bool useGridKernel = true;
const bool useSubWarpKernel = false;
const bool useThreadBlockKernel = false; //FIXME: Thread block kernel does not work for subWarp < 32 
const bool combineTwoSampleStores = true;

enum TransitKernelTypes {
  GridKernel = 1,
  ThreadBlockKernel = 2,
  SubWarpKernel = 3,
  IdentityKernel = 4,
  NumKernelTypes = 4
};

/**User Defined Functions**/
enum SamplingType {
  IndividualNeighborhood,
  CollectiveNeighborhood
};

enum OutputFormat {
  SampledVertices,
  AdjacencyMatrix
};

__constant__ char csrPartitionBuff[sizeof(CSRPartition)];

template<typename T, size_t CACHE_SIZE, bool ONDEMAND_CACHING, int STATIC_CACHE_SIZE>
struct CachedArray {
  const T* glArray;
  T* shArray;
  
  __device__
  T operator[](int id)
  {
    return at(id);
  }

  __device__
  T at(int id)
  {
    if (id >= CACHE_SIZE) {
      return glArray[id];
    }
    
    VertexID_t e;

    // if (false && COALESCE_GL_LOADS) {
    //   e = cachedEdges[id];
    //   if (e == -1)
    //     cachedEdges[id] = -2;

    //   int subWarpThreadIdx = threadIdx.x % LoadBalancing::LoadBalancingThreshold::SubWarpLevel;
    //   //int subWarp = threadIdx.x / LoadBalancing::LoadBalancingThreshold::SubWarpLevel;
    //   for (int i = subWarpThreadIdx; i < CACHE_SIZE; i += LoadBalancing::LoadBalancingThreshold::SubWarpLevel) {
    //     if (cachedEdges[i] == -2) {
    //       cachedEdges[i] = transitEdges[i];
    //     }
    //   }
      
    //   e = cachedEdges[id];
    // } else 
    {
      e = shArray[id];
      if (ONDEMAND_CACHING and id < STATIC_CACHE_SIZE)
        return e;

      if (ONDEMAND_CACHING and e == -1) {
        e = glArray[id];
        shArray[id] = e;
      }
    }

    return e;
  }
};

template<typename App>
__host__ __device__
EdgePos_t newNeighborsSize(int hop, EdgePos_t num_edges)
{
  return (App().stepSize(hop) == ALL_NEIGHBORS) ? num_edges : (EdgePos_t)App().stepSize(hop);
}

template<typename App>
__host__ __device__
EdgePos_t subWarpSizeAtStep(int step)
{
  if (step == -1)
    return 0;
  
  //SubWarpSize is set to next power of 2
  
  EdgePos_t x = App().stepSize(step);

  if (x && (!(x&(x-1)))) {
    return x;
  } 

  x--;
  x |= x >> 1;
  x |= x >> 2;
  x |= x >> 4;
  x |= x >> 8;
  x |= x >> 16;
  if (sizeof(EdgePos_t) == sizeof(uint64_t)) {
    //x |= x >> 32;
  }
  x++;

  return x;
}

template<typename App>
__host__ __device__
EdgePos_t stepSizeAtStep(int step)
{
  if (step == -1)
    return 0;

  if (App().samplingType() == IndividualNeighborhood) {
    EdgePos_t n = App().initialSampleSize(nullptr);
    for (int i = 0; i <= step; i++) {
      n = n * App().stepSize(i);
    }

    return n;
  } else if (App().samplingType() == CollectiveNeighborhood) {
    EdgePos_t n = 0;
    for (int i = 0; i <= step; i++) {
      n += App().stepSize(i);
    }

    return n;
  }

  return -1;
}

template<typename App>
__host__ __device__ int numberOfTransits(int step) {
  if (App().samplingType() == CollectiveNeighborhood) {
    return App().stepSize(step);
  } else if (App().samplingType() == IndividualNeighborhood) {
    return stepSizeAtStep<App>(step);
  }
  assert(false);
  return -1;
}

__host__ __device__ bool isValidSampledVertex(VertexID_t neighbor, VertexID_t InvalidVertex) 
{
  return neighbor != InvalidVertex && neighbor != -1;
}

enum TransitParallelMode {
  //Describes the execution mode of Transit Parallel.
  NextFuncExecution, //Execute the next function
  CollectiveNeighborhoodSize, //Compute size of collective neighborhood
  CollectiveNeighborhoodComputation, //Compute the collective neighborhood 
};

#define STORE_TRANSIT_INDEX false
template<class SamplingType, typename App, TransitParallelMode tpMode, int CollNeighStepSize>
__global__ void samplingKernel(const int step, GPUCSRPartition graph, const size_t threadsExecuted, const size_t currExecutionThreads,
                               const VertexID_t deviceFirstSample, const VertexID_t invalidVertex,
                               const VertexID_t* transitToSamplesKeys, const VertexID_t* transitToSamplesValues,
                               const size_t transitToSamplesSize, SamplingType* samples, const size_t NumSamples,
                               VertexID_t* samplesToTransitKeys, VertexID_t* samplesToTransitValues,
                               VertexID_t* finalSamples, const size_t finalSampleSize, EdgePos_t* sampleInsertionPositions,
                               EdgePos_t* sampleNeighborhoodSizes, EdgePos_t* sampleNeighborhoodPos, 
                               VertexID_t* collectiveNeighborhoodCSRRows, 
                               EdgePos_t* collectiveNeighborhoodCSRCols, hiprandState* randStates)
{
  EdgePos_t threadId = threadIdx.x + blockDim.x * blockIdx.x;
  //__shared__ VertexID newNeigbhors[N_THREADS];

  if (threadId >= currExecutionThreads)
    return;
  
  hiprandState* randState = &randStates[threadId];

  threadId += threadsExecuted;
  int stepSize;
  if (tpMode == NextFuncExecution) {
    stepSize = App().stepSize(step);
  } else if (tpMode == CollectiveNeighborhoodComputation) {
    stepSize = CollNeighStepSize;
  } else if (tpMode == CollectiveNeighborhoodSize) {
    stepSize = 1;
  }
  EdgePos_t transitIdx = threadId/App().stepSize(step);
  EdgePos_t transitNeighborIdx = threadId % App().stepSize(step);
  EdgePos_t numTransits = numberOfTransits<App>(step);

  VertexID_t sampleIdx = transitToSamplesValues[transitIdx];
  assert(sampleIdx < NumSamples);
  VertexID_t transit = transitToSamplesKeys[transitIdx];
  VertexID_t neighbor = invalidVertex;
  graph.device_csr = (CSRPartition*)&csrPartitionBuff[0];

  if (transit != invalidVertex) {
    assert(graph.device_csr->has_vertex(transit));

    EdgePos_t numTransitEdges = graph.device_csr->get_n_edges_for_vertex(transit);
    
    if (numTransitEdges != 0 && (tpMode == NextFuncExecution || tpMode == CollectiveNeighborhoodComputation)) {
      //Execute next in this mode only
      const CSR::Edge* transitEdges = graph.device_csr->get_edges(transit);
      const float* transitEdgeWeights = graph.device_csr->get_weights(transit);
      const float maxWeight = graph.device_csr->get_max_weight(transit);
      if (tpMode == NextFuncExecution) {
        neighbor = App().next(step, graph.device_csr, &transit, sampleIdx, &samples[(sampleIdx - deviceFirstSample)], maxWeight, transitEdges, transitEdgeWeights, 
                              numTransitEdges, transitNeighborIdx, randState);
      } else {
        int insertionPos = utils::atomicAdd(&sampleInsertionPositions[sampleIdx- deviceFirstSample], numTransitEdges);
        collectiveNeighborhoodCSRRows[(sampleIdx - deviceFirstSample)*App().initialSampleSize(nullptr) + 0] = insertionPos;

        for (int e = transitNeighborIdx; e < numTransitEdges; e += stepSize) {
          EdgePos_t pos = sampleNeighborhoodPos[(sampleIdx - deviceFirstSample)] + e + insertionPos%2;
          collectiveNeighborhoodCSRCols[pos] = transitEdges[e];
        }
      }
    } else if (tpMode == CollectiveNeighborhoodSize) {
      //Compute size of collective neighborhood for each sample.
      ::atomicAdd(&sampleNeighborhoodSizes[(sampleIdx - deviceFirstSample)], numTransitEdges);
    }
  }

  __syncwarp();
  if (tpMode == NextFuncExecution) {
    EdgePos_t insertionPos = 0;
    //TODO: templatize over hasExplicitTransits()
    if (step != App().steps() - 1) {
      //No need to store at last step
      if (App().hasExplicitTransits()) {
        VertexID_t newTransit = App().stepTransits(step+1, sampleIdx, samples[(sampleIdx - deviceFirstSample)], threadId%numTransits, randState);
        samplesToTransitValues[threadId] = newTransit != -1 ? newTransit : invalidVertex;
      } else {
        samplesToTransitValues[threadId] = neighbor != -1 ? neighbor : invalidVertex;
      }
      samplesToTransitKeys[threadId] = sampleIdx;
    }

    if (numberOfTransits<App>(step) > 1 && isValidSampledVertex(neighbor, invalidVertex)) {   
      //insertionPos = finalSampleSizeTillPreviousStep + transitNeighborIdx; //
      if (step == 0) {
        insertionPos = transitNeighborIdx;
      } else {
        size_t finalSampleSizeTillPreviousStep = 0;
        size_t neighborsToSampleAtStep = 1;
        for (int _s = 0; _s < step; _s++) {
          neighborsToSampleAtStep *= App().stepSize(_s);
          finalSampleSizeTillPreviousStep += neighborsToSampleAtStep;
        }
        insertionPos = finalSampleSizeTillPreviousStep + utils::atomicAdd(&sampleInsertionPositions[(sampleIdx - deviceFirstSample)], 1);
      }
    } else {
      insertionPos = step;
    }

    assert(finalSampleSize > 0);
    // if (insertionPos >= finalSampleSize) {
    //   printf("insertionPos %d finalSampleSize %ld sample %d\n", insertionPos, finalSampleSize, sampleIdx);
    // }
    assert(insertionPos < finalSampleSize);
    if (numberOfTransits<App>(step) == 1 and combineTwoSampleStores) {
      if (step % 2 == 1) {
        finalSamples[(sampleIdx - deviceFirstSample)*finalSampleSize + insertionPos - 1] = transit;
        if (isValidSampledVertex(neighbor, invalidVertex)) finalSamples[(sampleIdx - deviceFirstSample)*finalSampleSize + insertionPos] = neighbor;
      } else if (step == App().steps() - 1 && isValidSampledVertex(neighbor, invalidVertex)) {
        finalSamples[(sampleIdx - deviceFirstSample)*finalSampleSize + insertionPos] = neighbor;
      }
    }
    else {
      // if (STORE_TRANSIT_INDEX) {
      //   //Store Index of transit in each sample's output
      //   if (step == 0) {
      //     transitIndexInSample[threadId] = insertionPos;
      //   } else if (step != App().steps() - 1) {
      //     transitIndexInSample[threadId] = prevTransitIndexInSample[];
      //   }
      // }
      if (isValidSampledVertex(neighbor, invalidVertex))
        finalSamples[(sampleIdx - deviceFirstSample)*finalSampleSize + insertionPos] = neighbor;
    }
  }
}

template<class SampleType, typename App, int THREADS, bool COALESCE_CURAND_LOAD, bool HAS_EXPLICIT_TRANSITS>
__global__ void identityKernel(const int step, GPUCSRPartition graph, const VertexID_t deviceFirstSample, const VertexID_t invalidVertex,
                               const VertexID_t* transitToSamplesKeys, const VertexID_t* transitToSamplesValues,
                               const size_t transitToSamplesSize, SampleType* samples, const size_t NumSamples,
                               VertexID_t* samplesToTransitKeys, VertexID_t* samplesToTransitValues,
                               VertexID_t* finalSamples, const size_t finalSampleSize, EdgePos_t* sampleInsertionPositions,
                               hiprandState* randStates, const int* kernelTypeForTransit, int numTransits)
{
  __shared__ unsigned char shMemCuRand[sizeof(hiprandState)*THREADS];

  int threadId = threadIdx.x + blockDim.x * blockIdx.x;

  hiprandState* curandSrcPtr;

  if (COALESCE_CURAND_LOAD) {
    const int intsInRandState = sizeof(hiprandState)/sizeof(int);
    int* shStateBuff = (int*)&shMemCuRand[0];

    int* randStatesAsInts = (int*)randStates;
  
    for (int i = threadIdx.x; i < intsInRandState*blockDim.x; i += blockDim.x) {
      shStateBuff[i] = randStatesAsInts[i + blockDim.x*blockIdx.x];
    }

    __syncthreads();
    curandSrcPtr = (hiprandState*)(&shStateBuff[threadIdx.x*intsInRandState]);
  } else {
    curandSrcPtr = &randStates[threadId];
  }

  hiprandState* localRandState = curandSrcPtr;
  
  for (; threadId < transitToSamplesSize; threadId += gridDim.x * blockDim.x) {
    //__shared__ VertexID newNeigbhors[N_THREADS];
    EdgePos_t transitIdx;
    EdgePos_t transitNeighborIdx;
    VertexID_t transit;
    int kernelTy;
    bool continueExecution = true;

    continueExecution = threadId < transitToSamplesSize;
    
    int subWarpSize = subWarpSizeAtStep<App>(step);
    transitIdx = threadId/subWarpSize;
    transitNeighborIdx = threadId % subWarpSize;
    if (continueExecution && transitNeighborIdx == 0) {
      transit = transitToSamplesKeys[transitIdx];
      kernelTy = kernelTypeForTransit[transit];
    }

    transit = __shfl_sync(FULL_WARP_MASK, transit, 0, subWarpSize);
    kernelTy = __shfl_sync(FULL_WARP_MASK, kernelTy, 0, subWarpSize);

    continueExecution = continueExecution && transitNeighborIdx < App().stepSize(step);

    if ((useGridKernel && kernelTy == TransitKernelTypes::GridKernel && numTransits > 1) ||
        (useSubWarpKernel && kernelTy == TransitKernelTypes::SubWarpKernel && numTransits > 1) || 
        (useThreadBlockKernel && kernelTy == TransitKernelTypes::ThreadBlockKernel && numTransits > 1)) {
        continueExecution = false;
    }

    CSRPartition* csr = (CSRPartition*)&csrPartitionBuff[0];
    VertexID_t sampleIdx = -1;
    
    if (continueExecution && transitNeighborIdx == 0) {
      sampleIdx = transitToSamplesValues[transitIdx];
    }

    sampleIdx = __shfl_sync(FULL_WARP_MASK, sampleIdx, 0, subWarpSize);
    VertexID_t neighbor = invalidVertex;

    if (continueExecution and transit != invalidVertex) {
      EdgePos_t numTransitEdges = csr->get_n_edges_for_vertex(transit);
      
      if (numTransitEdges != 0) {
        const CSR::Edge* transitEdges = csr->get_edges(transit);
        const float* transitEdgeWeights = csr->get_weights(transit);
        const float maxWeight = csr->get_max_weight(transit);

        neighbor = App().next(step, csr, &transit, sampleIdx, &samples[sampleIdx - deviceFirstSample], maxWeight, transitEdges, transitEdgeWeights, 
                        numTransitEdges, transitNeighborIdx, localRandState);
  #if 0
        //search if neighbor has already been selected.
        //we can do that in register if required
        newNeigbhors[threadIdx.x] = neighbor;

        bool found = false;
        for (int i = 0; i < N_THREADS; i++) {
          if (newNeigbhors[i] == neighbor) {
            found = true;
            // break;
          }
        }

        __syncwarp();
        if (found) {
          neighbor = next(step, transit, sample, transitEdges, numTransitEdges, 
            transitNeighborIdx, randState);;
        }
  #endif
      }
    }

    __syncwarp();
  
  if (continueExecution) {
    if (step != App().steps() - 1) {
      //No need to store at last step
      if (HAS_EXPLICIT_TRANSITS) {
        VertexID_t newTransit = App().stepTransits(step + 1, sampleIdx, samples[sampleIdx - deviceFirstSample], transitIdx, localRandState);
        samplesToTransitValues[threadId] = newTransit != -1 ? newTransit : invalidVertex;
      } else {
        samplesToTransitValues[threadId] = neighbor != -1 ? neighbor : invalidVertex;;
      }
      samplesToTransitKeys[threadId] = sampleIdx;
    }
  }

  __syncwarp();
  //FIXME: in deepwalk if there is an invalid vertex at step k, it will not store the
  //transits of step k -1 due to coalescing the stores. 
  EdgePos_t finalSampleSizeTillPreviousStep = 0;
  EdgePos_t neighborsToSampleAtStep = 1;
  EdgePos_t insertionPos = 0; 
  if (numTransits > 1) {    
    if (step == 0) {
      insertionPos = transitNeighborIdx;
    } else {
      for (int _s = 0; _s < step; _s++) {
        neighborsToSampleAtStep *= App().stepSize(_s);
        finalSampleSizeTillPreviousStep += neighborsToSampleAtStep;
      }
      EdgePos_t insertionStartPosForTransit = 0;

      if (threadIdx.x % subWarpSize == 0) {
          insertionStartPosForTransit = utils::atomicAdd(&sampleInsertionPositions[sampleIdx - deviceFirstSample], App().stepSize(step));
      }
      insertionStartPosForTransit = __shfl_sync(FULL_WARP_MASK, insertionStartPosForTransit, 0, subWarpSize);
      insertionPos = finalSampleSizeTillPreviousStep + insertionStartPosForTransit + transitNeighborIdx;
    }
  } else {
    insertionPos = step;
  }

  __syncwarp();
  if (continueExecution) {
    if (combineTwoSampleStores && numTransits == 1) {
      //TODO: We can combine stores even when numberOfTransits<App>(step) > 1
      if (step % 2 == 1) {
        int2 *ptr = (int2*)&finalSamples[(sampleIdx - deviceFirstSample)*finalSampleSize + insertionPos - 1];
        int2 res;
        res.x = transit;
        res.y = neighbor;
        *ptr = res;
        //finalSamples[sample*finalSampleSize + insertionPos] = neighbor;
      } else if (step == App().steps() - 1) {
        finalSamples[(sampleIdx - deviceFirstSample)*finalSampleSize + insertionPos] = neighbor;
      }
    } else {
      if (isValidSampledVertex(neighbor, invalidVertex))
        finalSamples[(sampleIdx - deviceFirstSample)*finalSampleSize + insertionPos] = neighbor;
    }
  }
  //TODO: We do not need atomic instead store indices of transit in another array,
  //wihch can be accessed based on sample and transitIdx.
  }
}

template<class SampleType, typename App, int THREADS, int CACHE_SIZE, bool CACHE_EDGES, bool CACHE_WEIGHTS, bool COALESCE_GL_LOADS, int TRANSITS_PER_THREAD, bool COALESCE_CURAND_LOAD>
__global__ void subWarpKernel(const int step, GPUCSRPartition graph, const VertexID_t invalidVertex,
                              const VertexID_t* transitToSamplesKeys, const VertexID_t* transitToSamplesValues,
                              const size_t transitToSamplesSize, SampleType* samples, const size_t NumSamples,
                              VertexID_t* samplesToTransitKeys, VertexID_t* samplesToTransitValues,
                              VertexID_t* finalSamples, const size_t finalSampleSize, EdgePos_t* sampleInsertionPositions,
                              hiprandState* randStates, const int* kernelTypeForTransit, const VertexID_t* subWarpKernelTBPositions, 
                              const EdgePos_t subWarpKernelTBPositionsNum)
{  
  // __shared__ unsigned char shMemAlloc[sizeof(hiprandState)*THREADS];
  // __shared__ EdgePos_t shSubWarpPositions[SUBWARPS_IN_TB*TRANSITS_PER_THREAD];
  const int SUBWARPS_IN_TB = THREADS/LoadBalancing::LoadBalancingThreshold::SubWarpLevel;
  const int EDGE_CACHE_SIZE = (CACHE_EDGES ? CACHE_SIZE * sizeof(CSR::Edge) : 0);
  const int WEIGHT_CACHE_SIZE = (CACHE_WEIGHTS ? CACHE_SIZE * sizeof(float) : 0);
  const int TOTAL_CACHE_SIZE = MAX(WEIGHT_CACHE_SIZE + EDGE_CACHE_SIZE, 1); 
  const int CACHE_SIZE_PER_SUBWARP = CACHE_SIZE/SUBWARPS_IN_TB;

  union unionShMem {
    struct {
      EdgePos_t shSubWarpPositions[SUBWARPS_IN_TB*TRANSITS_PER_THREAD];
      unsigned char edgeAndWeightCache[TOTAL_CACHE_SIZE];
    };
    unsigned char shMemAlloc[sizeof(hiprandState)*THREADS];
  };
  __shared__ unionShMem shMem;
  
  const int threadId = threadIdx.x + blockDim.x * blockIdx.x;

  const int subWarpThreadIdx = threadId % LoadBalancing::LoadBalancingThreshold::SubWarpLevel;
  const int subWarp = threadId / LoadBalancing::LoadBalancingThreshold::SubWarpLevel;
  const int subWarpIdxInTB = threadIdx.x/LoadBalancing::LoadBalancingThreshold::SubWarpLevel;
  const int startSubWarpIdxInTB = (blockIdx.x*blockDim.x)/LoadBalancing::LoadBalancingThreshold::SubWarpLevel;

  EdgePos_t* edgesInShMem = (EdgePos_t*) (CACHE_EDGES ? &shMem.edgeAndWeightCache[CACHE_SIZE_PER_SUBWARP*subWarpIdxInTB] : nullptr);
  float* edgeWeightsInShMem = (float*) (CACHE_WEIGHTS ? (&shMem.edgeAndWeightCache[EDGE_CACHE_SIZE + CACHE_SIZE_PER_SUBWARP*subWarpIdxInTB]): nullptr);
  bool* globalLoadBV = nullptr;

  hiprandState* curandSrcPtr;

  if (COALESCE_CURAND_LOAD) {
    const int intsInRandState = sizeof(hiprandState)/sizeof(int);
    int* shStateBuff = (int*)&shMem.shMemAlloc[0];

    int* randStatesAsInts = (int*)randStates;
  
    for (int i = threadIdx.x; i < intsInRandState*blockDim.x; i += blockDim.x) {
      shStateBuff[i] = randStatesAsInts[i + blockDim.x*blockIdx.x];
    }

    __syncthreads();
    curandSrcPtr = (hiprandState*)(&shStateBuff[threadIdx.x*intsInRandState]);
  } else {
    curandSrcPtr = &randStates[threadId];
  }

  hiprandState localRandState = *curandSrcPtr;
  
  for (int _subWarpIdx = threadIdx.x; _subWarpIdx < SUBWARPS_IN_TB * TRANSITS_PER_THREAD; _subWarpIdx += blockDim.x) {
    if (_subWarpIdx + startSubWarpIdxInTB * TRANSITS_PER_THREAD >= subWarpKernelTBPositionsNum) {
      continue;
    }
    shMem.shSubWarpPositions[_subWarpIdx] = subWarpKernelTBPositions[_subWarpIdx + startSubWarpIdxInTB * TRANSITS_PER_THREAD];
  }

  __syncthreads();
  bool invalidateCache;
  VertexID_t currTransit = invalidVertex;

  invalidateCache = true;
  EdgePos_t numTransitEdges;
  CSR::Edge* glTransitEdges;
  float* glTransitEdgeWeights;
  float maxWeight;

  for (int transitI = 0; transitI < TRANSITS_PER_THREAD; transitI++) {
    EdgePos_t subWarpIdx = TRANSITS_PER_THREAD * subWarp + transitI;
    if (subWarpIdx >= subWarpKernelTBPositionsNum) {
      continue;
    }

    EdgePos_t transitStartPos = shMem.shSubWarpPositions[subWarpIdxInTB * TRANSITS_PER_THREAD + transitI];
    EdgePos_t transitIdx = transitStartPos + subWarpThreadIdx;
    EdgePos_t transitNeighborIdx = 0;
    VertexID_t transit = transitIdx < NumSamples ? transitToSamplesKeys[transitIdx] : -1;

    VertexID_t firstThreadTransit = __shfl_sync(FULL_WARP_MASK, transit, 0, LoadBalancing::LoadBalancingThreshold::SubWarpLevel);
    __syncwarp();

    invalidateCache = currTransit != firstThreadTransit;
    currTransit = firstThreadTransit;

    CSRPartition* csr = (CSRPartition*)&csrPartitionBuff[0];
    
    int tmpReadVertexData;

    if (invalidateCache) {
      const CSR::Vertex* transitVertex = csr->get_vertex(currTransit);
      if (subWarpThreadIdx < sizeof(CSR::Vertex)/sizeof(int)) {
        tmpReadVertexData = ((const int*)transitVertex)[subWarpThreadIdx];
      }
    }
    
    __syncwarp();

    const EdgePos_t startEdgeIdx = __shfl_sync(FULL_WARP_MASK, tmpReadVertexData, 1, LoadBalancing::LoadBalancingThreshold::SubWarpLevel);
    const EdgePos_t endEdgeIdx = __shfl_sync(FULL_WARP_MASK, tmpReadVertexData, 2, LoadBalancing::LoadBalancingThreshold::SubWarpLevel);
    
    if (invalidateCache) {
      int maxWeightBuff = __shfl_sync(FULL_WARP_MASK, tmpReadVertexData, 3, LoadBalancing::LoadBalancingThreshold::SubWarpLevel);      
      maxWeight = *((float*)&maxWeightBuff);
      numTransitEdges = (endEdgeIdx != -1) ? (endEdgeIdx - startEdgeIdx + 1) : 0; 
     
      glTransitEdges = (CSR::Edge*)((startEdgeIdx != -1) ? csr->get_edges() + startEdgeIdx : nullptr);
      glTransitEdgeWeights = (float*)((startEdgeIdx != -1) ? csr->get_weights() + startEdgeIdx : nullptr);
    }

    if (false) {
      //shMem.edgeAndWeightCache[threadIdx.x%32] = numTransitEdges + (int32_t)maxWeight + (int32_t)glTransitEdges + (int32_t)glTransitEdgeWeights;
      continue;
    }

    if (CACHE_EDGES && invalidateCache) {
      for (int e = subWarpThreadIdx; e < min((EdgePos_t)CACHE_SIZE_PER_SUBWARP, numTransitEdges); 
           e += LoadBalancing::LoadBalancingThreshold::SubWarpLevel) {
        edgesInShMem[e] = -1;
      }
    }

    if (CACHE_WEIGHTS && invalidateCache) {
      for (int e = subWarpThreadIdx; e < min((EdgePos_t)CACHE_SIZE_PER_SUBWARP, numTransitEdges); 
           e += LoadBalancing::LoadBalancingThreshold::SubWarpLevel) {
        edgeWeightsInShMem[e] = -1;
      }
    }

    __syncwarp();

    if (firstThreadTransit != transit)
      continue;


    //assert(kernelTypeForTransit[firstThreadTransit] == TransitKernelTypes::SubWarpKernel);
    VertexID_t sampleIdx = transitToSamplesValues[transitIdx];
    assert(sampleIdx < NumSamples);
    VertexID_t neighbor = invalidVertex;

   
    assert(csr->has_vertex(transit));
    // CachedArray<CSR::Edge, CACHE_SIZE, ONDEMAND_CACHING, STATIC_CACHE_SIZE> cachedEdges = {shMem.glTransitEdges, edgesInShMem};
    if (numTransitEdges != 0) {
      assert(false);//TODO: Disable for now.
      // neighbor = App().template nextCached<SampleType, CACHE_SIZE_PER_SUBWARP, CACHE_EDGES, CACHE_WEIGHTS, COALESCE_GL_LOADS, false, 0>(step, transit, sampleIdx, &samples[sampleIdx], maxWeight, 
      //                                                                               cachedEdges, glTransitEdgeWeights, 
      //                                                                               numTransitEdges, transitNeighborIdx, &localRandState,
      //                                                                               edgeWeightsInShMem,
      //                                                                               globalLoadBV);
    }

    // __syncwarp();

    //EdgePos_t totalSizeOfSample = stepSizeAtStep<App>(step - 1);

    if (step != App().steps() - 1) {
      //No need to store at last step
      samplesToTransitKeys[transitIdx] = sampleIdx;
      if (App().hasExplicitTransits()) {
        VertexID_t transit = App().stepTransits(step, sampleIdx, samples[sampleIdx], transitIdx, &localRandState);
        samplesToTransitValues[threadId] = transit;
      } else {
        samplesToTransitValues[threadId] = neighbor;
      }
    }
    
    EdgePos_t insertionPos = 0; 
    if (false && numberOfTransits<App>(step) > 1) {    
      insertionPos = utils::atomicAdd(&sampleInsertionPositions[sampleIdx], 1);
    } else {
      insertionPos = step;
    }

 
    assert(finalSampleSize > 0);
    assert(insertionPos < finalSampleSize);
    if (combineTwoSampleStores) {
      if (step % 2 == 1) {
        int2 *ptr = (int2*)&finalSamples[sampleIdx*finalSampleSize + insertionPos - 1];
        int2 res;
        res.x = transit;
        res.y = neighbor;
        *ptr = res;
        //finalSamples[sample*finalSampleSize + insertionPos] = neighbor;
      } else if (step == App().steps() - 1) {
        finalSamples[sampleIdx*finalSampleSize + insertionPos] = neighbor;
      }
    } else {
      finalSamples[sampleIdx*finalSampleSize + insertionPos] = neighbor;
    }
    
    //TODO: We do not need atomic instead store indices of transit in another array,
    //wich can be accessed based on sample and transitIdx.
  }
}

template<class SampleType, typename App, int THREADS, int CACHE_SIZE, bool CACHE_EDGES, bool CACHE_WEIGHTS, int TRANSITS_PER_THREAD, bool ONDEMAND_CACHING, int STATIC_CACHE_SIZE, int SUB_WARP_SIZE>
__global__ void threadBlockKernel(const int step, GPUCSRPartition graph, const VertexID_t deviceFirstSample, 
                           const VertexID_t invalidVertex,
                           const VertexID_t* transitToSamplesKeys, const VertexID_t* transitToSamplesValues,
                           const size_t transitToSamplesSize, SampleType* samples, const size_t NumSamples,
                           VertexID_t* samplesToTransitKeys, VertexID_t* samplesToTransitValues,
                           VertexID_t* finalSamples, const size_t finalSampleSize, EdgePos_t* sampleInsertionPositions,
                           hiprandState* randStates, const int* kernelTypeForTransit, const VertexID_t* threadBlockKernelPositions, 
                           const EdgePos_t threadBlockKernelPositionsNum, int totalThreadBlocks,
                           int numTransitsAtStepPerSample, int finalSampleSizeTillPreviousStep)
{
  #define EDGE_CACHE_SIZE (CACHE_EDGES ? CACHE_SIZE*sizeof(CSR::Edge) : 0)
  #define WEIGHT_CACHE_SIZE (CACHE_WEIGHTS ? CACHE_SIZE*sizeof(float) : 0)
  #define CURAND_SHMEM_SIZE (sizeof(hiprandState)*THREADS)
  #define NUM_THREAD_GROUPS (THREADS / LoadBalancing::LoadBalancingThreshold::BlockLevel)

  union unionShMem {
    struct {
      unsigned char edgeAndWeightCache[EDGE_CACHE_SIZE*NUM_THREAD_GROUPS+WEIGHT_CACHE_SIZE*NUM_THREAD_GROUPS];
      // VertexID_t transitForSubWarp[NUM_THREAD_GROUPS];
      EdgePos_t mapStartPos[NUM_THREAD_GROUPS][TRANSITS_PER_THREAD];
      EdgePos_t subWarpTransits[NUM_THREAD_GROUPS][TRANSITS_PER_THREAD][LoadBalancing::LoadBalancingThreshold::BlockLevel/SUB_WARP_SIZE];
      EdgePos_t subWarpSampleIdx[NUM_THREAD_GROUPS][TRANSITS_PER_THREAD][LoadBalancing::LoadBalancingThreshold::BlockLevel/SUB_WARP_SIZE];
      unsigned char transitVertices[NUM_THREAD_GROUPS][TRANSITS_PER_THREAD*sizeof(CSR::Vertex)];
    };
    unsigned char shMemAlloc[sizeof(hiprandState)*THREADS];
  };
  __shared__ unionShMem shMem;
    
  CSR::Edge* edgesInShMem = CACHE_EDGES ? (CSR::Edge*)(&shMem.edgeAndWeightCache[0] + EDGE_CACHE_SIZE*(threadIdx.x/LoadBalancing::LoadBalancingThreshold::BlockLevel)) : nullptr;
  float* edgeWeightsInShMem = CACHE_WEIGHTS ? (float*)&shMem.edgeAndWeightCache[EDGE_CACHE_SIZE] : nullptr;
  
  int threadId = threadIdx.x + blockDim.x * blockIdx.x;
  const int stepSize = App().stepSize(step);
  hiprandState* curandSrcPtr;

  const int subWarpSize = SUB_WARP_SIZE;

  if (true) {
    const int intsInRandState = sizeof(hiprandState)/sizeof(int);
    int* shStateBuff = (int*)&shMem.shMemAlloc[0];

    int* randStatesAsInts = (int*)randStates;
    
    //Load hiprand only for the number of threads that are going to do sampling in this warp
    for (int i = threadIdx.x; i < intsInRandState*(blockDim.x/subWarpSize)*stepSize; i += blockDim.x) {
      shStateBuff[i] = randStatesAsInts[i + blockDim.x*blockIdx.x];
    }

    __syncthreads();
    if (threadIdx.x % subWarpSize < stepSize) {
      //Load hiprand only for the threads that are going to do sampling.
      // int ld = threadIdx.x - (threadIdx.x/subWarpSize)*(subWarpSize-stepSize);
      curandSrcPtr = (hiprandState*)(&shStateBuff[threadIdx.x*intsInRandState]);
    }
  } else {
    curandSrcPtr = &randStates[threadId];
  }

  hiprandState localRandState = (threadIdx.x % subWarpSize < stepSize)? *curandSrcPtr: hiprandState();
  
  CSRPartition* csr = (CSRPartition*)&csrPartitionBuff[0];
  for (int fullBlockIdx = blockIdx.x; fullBlockIdx < totalThreadBlocks; fullBlockIdx += gridDim.x) {
    EdgePos_t transitIdx = 0;
    static_assert(NUM_THREAD_GROUPS * TRANSITS_PER_THREAD <= THREADS);
    int fullWarpIdx = (threadIdx.x + fullBlockIdx * blockDim.x)/LoadBalancing::LoadBalancingThreshold::BlockLevel;

    if (threadIdx.x < NUM_THREAD_GROUPS * TRANSITS_PER_THREAD) {
      const int warpIdx = threadIdx.x/TRANSITS_PER_THREAD;
      const int transitIdx = threadIdx.x%TRANSITS_PER_THREAD;
      const int __fullWarpIdx = warpIdx + (fullBlockIdx * blockDim.x)/LoadBalancing::LoadBalancingThreshold::BlockLevel;

      if (TRANSITS_PER_THREAD * __fullWarpIdx + transitIdx < threadBlockKernelPositionsNum) {
        shMem.mapStartPos[warpIdx][transitIdx] = threadBlockKernelPositions[TRANSITS_PER_THREAD * __fullWarpIdx + transitIdx];
      } else {
        shMem.mapStartPos[warpIdx][transitIdx] = -1;
      }
    }
  
    __syncthreads();
    
    const int NUM_SUBWARPS_IN_TB = NUM_THREAD_GROUPS * (LoadBalancing::LoadBalancingThreshold::BlockLevel/SUB_WARP_SIZE);
    static_assert(NUM_SUBWARPS_IN_TB * TRANSITS_PER_THREAD <= THREADS);
    
    if (threadIdx.x < NUM_SUBWARPS_IN_TB * TRANSITS_PER_THREAD) {
      //Coalesce loads of transits per sub-warp by loading transits for all sub-warps in one warp.
      //FIXME: Fix this when SUB_WARP_SIZE < 32
      int subWarpIdx = threadIdx.x / TRANSITS_PER_THREAD;
      int transitI = threadIdx.x % TRANSITS_PER_THREAD;
      transitIdx = shMem.mapStartPos[subWarpIdx][transitI];
      //TODO: Specialize this for subWarpSize = 1.
      VertexID_t transit = invalidVertex;
      if (transitIdx != -1) {
        transit = transitToSamplesKeys[transitIdx];
        shMem.subWarpSampleIdx[subWarpIdx][transitI][0] = transitToSamplesValues[transitIdx];
      }
      shMem.subWarpTransits[subWarpIdx][transitI][0] = transit;
    }

    __syncthreads();

    const int threadsToLoadTransit = sizeof(CSR::Vertex)/sizeof(int);
    if (threadIdx.x < threadsToLoadTransit * NUM_SUBWARPS_IN_TB * TRANSITS_PER_THREAD) {
      //Load transit Vertex Object in a coalesced manner
      //TODO: Fix this for subwarpsize < 32
      int transitI = (threadIdx.x / (threadsToLoadTransit)) % TRANSITS_PER_THREAD;
      int subWarpIdx = threadIdx.x / (TRANSITS_PER_THREAD * threadsToLoadTransit);
      VertexID transit = shMem.subWarpTransits[subWarpIdx][transitI][0];
      if (transit != invalidVertex) {
        const CSR::Vertex* transitVertex = csr->get_vertices() + transit;
        int tid = threadIdx.x % threadsToLoadTransit;
        int data = ((const int*)transitVertex)[tid];
        *(((int*)&shMem.transitVertices[subWarpIdx][transitI * sizeof(CSR::Vertex)]) + tid) = data;
      }
    }
    
    __syncthreads();

    for (int transitI = 0; transitI < TRANSITS_PER_THREAD; transitI++) {
      int threadBlockWarpIdx = threadIdx.x / subWarpSize;
      //TODO: Support this for SubWarp != 32

      if (TRANSITS_PER_THREAD * fullWarpIdx + transitI >= threadBlockKernelPositionsNum) {
        continue;
      }

      __syncwarp(); //TODO: Add mask based on subwarp
      VertexID_t transit = -1;
      bool invalidateCache = false;
      if (threadIdx.x % subWarpSize == 0) {
        invalidateCache = shMem.subWarpTransits[threadBlockWarpIdx][transitI][0] != transit || transitI == 0;
      }
      
      invalidateCache = __shfl_sync(FULL_WARP_MASK, invalidateCache, 0, subWarpSize);

      transit = shMem.subWarpTransits[threadBlockWarpIdx][transitI][0];
      if (transit == invalidVertex) 
        continue;
      
      __syncwarp();

      // assert (transit != invalidVertex);
      CSR::Vertex* shMemTransitVertex = ((CSR::Vertex*)(&shMem.transitVertices[threadBlockWarpIdx][transitI * sizeof(CSR::Vertex)]));
      EdgePos_t numEdgesInShMem = shMemTransitVertex->num_edges();
      const CSR::Edge* glTransitEdges = (CSR::Edge*)csr->get_edges() + shMemTransitVertex->get_start_edge_idx();
      // assert(glTransitEdges == csr->get_edges(transit));
      // assert(glTransitEdges != nullptr);
      // assert(numEdgesInShMem == csr->get_n_edges_for_vertex(transit));
      // assert(shMemTransitVertex->get_start_edge_idx() + numEdgesInShMem <= csr->get_n_edges());
      const float* glTransitEdgeWeights = (float*)(CSR::Edge*)csr->get_weights() + shMemTransitVertex->get_start_edge_idx();
      float maxWeight = shMemTransitVertex->get_max_weight();
      
      if (CACHE_EDGES && invalidateCache) {
        for (int i = threadIdx.x%LoadBalancing::LoadBalancingThreshold::BlockLevel; 
             i < min(CACHE_SIZE, numEdgesInShMem); 
             i += LoadBalancing::LoadBalancingThreshold::BlockLevel) {
          if (ONDEMAND_CACHING) {
            if (i < STATIC_CACHE_SIZE)
              edgesInShMem[i] = glTransitEdges[i];
            else 
              edgesInShMem[i] = -1;
          } else {
            edgesInShMem[i] = glTransitEdges[i];
          }
        }
      }

      __syncwarp();
  
      // if (CACHE_WEIGHTS && shMem.invalidateCache) {
      //   for (int i = threadIdx.x; i < min(CACHE_SIZE, numEdgesInShMem); i += blockDim.x) {
      //     edgeWeightsInShMem[i] = (ONDEMAND_CACHING) ? -1 : glTransitEdgeWeights[i];
      //   }
      // }
      

      bool continueExecution = true;
      
      if (true){// || transit == shMem.transitForSubWarp[threadBlockWarpIdx]) {
        //A thread will run next only when it's transit is same as transit of the threadblock.
        transitIdx = shMem.mapStartPos[threadBlockWarpIdx][transitI] + threadIdx.x/subWarpSize; //threadId/stepSize(step);
        VertexID_t transitNeighborIdx = threadIdx.x % subWarpSize;
        VertexID_t sampleIdx = shMem.subWarpSampleIdx[threadBlockWarpIdx][transitI][0];

        // if (threadIdx.x % subWarpSize == 0) {
        //   sampleIdx = transitToSamplesValues[transitIdx];
        // }
        
        // sampleIdx = __shfl_sync(FULL_WARP_MASK, sampleIdx, 0, subWarpSize);

        continueExecution = (transitNeighborIdx < stepSize); 
       
        // assert (kernelTypeForTransit[transit] == TransitKernelTypes::GridKernel);
        
        //TODO: Set this based on the input template parameters.
        typedef CachedArray<CSR::Edge, CACHE_SIZE, ONDEMAND_CACHING, STATIC_CACHE_SIZE> CachedEdges;
        typedef CachedArray<float, 0, false, 0> CachedWeights;

        CachedEdges cachedEdges = {glTransitEdges, edgesInShMem};
        CachedWeights cachedWeights = {glTransitEdgeWeights, edgeWeightsInShMem};

        VertexID_t neighbor = invalidVertex;
        
        if (numEdgesInShMem > 0 && continueExecution) {
          neighbor = App().template next<SampleType, CachedEdges, CachedWeights>(step, csr, &transit, sampleIdx, &samples[sampleIdx-deviceFirstSample], maxWeight, 
                                                                                 cachedEdges, cachedWeights, numEdgesInShMem, transitNeighborIdx, &localRandState);
        }

        if (continueExecution) {
          if (step != App().steps() - 1) {
            //No need to store at last step
            samplesToTransitKeys[transitIdx] = sampleIdx; //TODO: Update this for khop to transitIdx + transitNeighborIdx
            if (App().hasExplicitTransits()) {
              VertexID_t newTransit = App().stepTransits(step, sampleIdx, samples[sampleIdx-deviceFirstSample], transitIdx, &localRandState);
              samplesToTransitValues[transitIdx] = newTransit != -1 ? newTransit : invalidVertex;
            } else {
              samplesToTransitValues[transitIdx] = neighbor != -1 ? neighbor : invalidVertex;
            }
          }
        }
        if (true) {
          EdgePos_t insertionPos = transitNeighborIdx; 
          if (numTransitsAtStepPerSample > 1) {
            if (step == 0) {
              insertionPos = transitNeighborIdx;
            } else {             
              EdgePos_t insertionStartPosForTransit = 0;
              //FIXME: 
              if (isValidSampledVertex(neighbor, invalidVertex) && threadIdx.x % subWarpSize == 0) {
                insertionStartPosForTransit = utils::atomicAdd(&sampleInsertionPositions[sampleIdx-deviceFirstSample], stepSize);
              }
              insertionStartPosForTransit = __shfl_sync(FULL_WARP_MASK, insertionStartPosForTransit, 0, subWarpSize);
              insertionPos = finalSampleSizeTillPreviousStep + insertionStartPosForTransit + transitNeighborIdx;
            }
          } else {
            insertionPos = step;
          }

          if (continueExecution)
          {
            assert(insertionPos < finalSampleSize);
          }
            

          if (combineTwoSampleStores && numTransitsAtStepPerSample == 1) {
            if (step % 2 == 1) {
              int2 *ptr = (int2*)&finalSamples[(sampleIdx-deviceFirstSample)*finalSampleSize + insertionPos - 1];
              int2 res;
              res.x = transit;
              res.y = neighbor;
              *ptr = res;
              //finalSamples[sample*finalSampleSize + insertionPos] = neighbor;
            } else if (step == App().steps() - 1) {
              finalSamples[(sampleIdx-deviceFirstSample)*finalSampleSize + insertionPos] = neighbor;
            }
          } else {
            if (continueExecution && isValidSampledVertex(neighbor, invalidVertex))
              finalSamples[(sampleIdx-deviceFirstSample)*finalSampleSize + insertionPos] = neighbor;
          }
          //TODO: We do not need atomic instead store indices of transit in another array,
          //wich can be accessed based on sample and transitIdx.
        }
      }
    }
  }
}

template<class SampleType, typename App, int THREADS, int CACHE_SIZE, bool CACHE_EDGES, bool CACHE_WEIGHTS, bool COALESCE_GL_LOADS, int TRANSITS_PER_THREAD, 
bool COALESCE_CURAND_LOAD, bool ONDEMAND_CACHING, int STATIC_CACHE_SIZE, int SUB_WARP_SIZE>
__global__ void gridKernel(const int step, GPUCSRPartition graph, const VertexID_t deviceFirstSample, 
                           const VertexID_t invalidVertex,
                           const VertexID_t* transitToSamplesKeys, const VertexID_t* transitToSamplesValues,
                           const size_t transitToSamplesSize, SampleType* samples, const size_t NumSamples,
                           VertexID_t* samplesToTransitKeys, VertexID_t* samplesToTransitValues,
                           VertexID_t* finalSamples, const size_t finalSampleSize, EdgePos_t* sampleInsertionPositions,
                           hiprandState* randStates, const int* kernelTypeForTransit, const VertexID_t* gridKernelTBPositions, 
                           const EdgePos_t gridKernelTBPositionsNum, int totalThreadBlocks, int numTransitsPerStepForSample,
                           int finalSampleSizeTillPreviousStep)
{
  #define EDGE_CACHE_SIZE (CACHE_EDGES ? CACHE_SIZE*sizeof(CSR::Edge) : 0)
  #define WEIGHT_CACHE_SIZE (CACHE_WEIGHTS ? CACHE_SIZE*sizeof(float) : 0)
  #define CURAND_SHMEM_SIZE (sizeof(hiprandState)*THREADS)
  // #define COALESCE_GL_LOADS_SHMEM_SIZE ()

  union unionShMem {
    struct {
      unsigned char edgeAndWeightCache[EDGE_CACHE_SIZE+WEIGHT_CACHE_SIZE];
      bool invalidateCache;
      VertexID_t transitForTB;
      EdgePos_t mapStartPos[TRANSITS_PER_THREAD];
      EdgePos_t subWarpTransits[TRANSITS_PER_THREAD][THREADS/SUB_WARP_SIZE];
      EdgePos_t subWarpSampleIdx[TRANSITS_PER_THREAD][THREADS/SUB_WARP_SIZE];
      unsigned char transitVertices[TRANSITS_PER_THREAD*sizeof(CSR::Vertex)];
    };
    unsigned char shMemAlloc[sizeof(hiprandState)*THREADS];
  };
  __shared__ unionShMem shMem;
  
  //__shared__ bool globalLoadBV[COALESCE_GL_LOADS ? CACHE_SIZE : 1];
  
  CSR::Edge* edgesInShMem = CACHE_EDGES ? (CSR::Edge*)&shMem.edgeAndWeightCache[0] : nullptr;
  float* edgeWeightsInShMem = CACHE_WEIGHTS ? (float*)&shMem.edgeAndWeightCache[EDGE_CACHE_SIZE] : nullptr;
  
  int threadId = threadIdx.x + blockDim.x * blockIdx.x;
  
  hiprandState* curandSrcPtr;
  const int stepSize = App().stepSize(step);

  const int subWarpSize = SUB_WARP_SIZE;

  if (COALESCE_CURAND_LOAD) {
    const int intsInRandState = sizeof(hiprandState)/sizeof(int);
    int* shStateBuff = (int*)&shMem.shMemAlloc[0];

    int* randStatesAsInts = (int*)randStates;
    
    //Load hiprand only for the number of threads that are going to do sampling in this warp
    for (int i = threadIdx.x; i < intsInRandState*(blockDim.x/subWarpSize)*stepSize; i += blockDim.x) {
      shStateBuff[i] = randStatesAsInts[i + blockDim.x*blockIdx.x];
    }

    __syncthreads();
    if (threadIdx.x % subWarpSize < stepSize) {
      //Load hiprand only for the threads that are going to do sampling.
      // int ld = threadIdx.x - (threadIdx.x/subWarpSize)*(subWarpSize-stepSize);
      curandSrcPtr = (hiprandState*)(&shStateBuff[threadIdx.x*intsInRandState]);
    }
  } else {
    curandSrcPtr = &randStates[threadId];
  }

  __syncwarp();

  hiprandState localRandState = (threadIdx.x % subWarpSize < stepSize)? *curandSrcPtr: hiprandState();
  //hiprand_init(threadId, 0,0, &localRandState);
  
  CSRPartition* csr = (CSRPartition*)&csrPartitionBuff[0];
  for (int fullBlockIdx = blockIdx.x; fullBlockIdx < totalThreadBlocks; fullBlockIdx += gridDim.x) {
    EdgePos_t transitIdx = 0;
    if (threadIdx.x < TRANSITS_PER_THREAD) {
      if (TRANSITS_PER_THREAD * fullBlockIdx + threadIdx.x < gridKernelTBPositionsNum) {
        shMem.mapStartPos[threadIdx.x] = gridKernelTBPositions[TRANSITS_PER_THREAD * fullBlockIdx + threadIdx.x];
      } else {
        shMem.mapStartPos[threadIdx.x] = -1;
      }
    }
    
    __syncthreads();
    if (threadIdx.x < THREADS/SUB_WARP_SIZE * TRANSITS_PER_THREAD) {
      //Coalesce loads of transits per sub-warp by loading transits for all sub-warps in one warp.
      // Assign THREADS/SUB_WARP_SIZE threads to each Transit in TRANSITS_PER_THREAD
      // static_assert ((THREADS/SUB_WARP_SIZE * TRANSITS_PER_THREAD) < THREADS);
      int transitI = threadIdx.x / (THREADS/SUB_WARP_SIZE);// * TRANSITS_PER_THREAD);
      transitIdx = shMem.mapStartPos[transitI] + threadIdx.x % (THREADS/SUB_WARP_SIZE);
      //TODO: Specialize this for subWarpSizez = 1.
      VertexID_t transit = invalidVertex;
      if (transitIdx != -1) {
        transit = transitToSamplesKeys[transitIdx];
        shMem.subWarpSampleIdx[transitI][threadIdx.x%(THREADS/SUB_WARP_SIZE)] = transitToSamplesValues[transitIdx];
      }
      shMem.subWarpTransits[transitI][threadIdx.x%(THREADS/SUB_WARP_SIZE)] = transit;
    }

    __syncwarp();
    __syncthreads();
    const int threadsToLoadTransit = sizeof(CSR::Vertex)/sizeof(int);
    if (threadIdx.x < threadsToLoadTransit * TRANSITS_PER_THREAD) {
      //Load Transit Vertex of first subwarp in a Coalesced manner
      int transitI = threadIdx.x / threadsToLoadTransit;
      VertexID transit = shMem.subWarpTransits[transitI][0];
      if (transit != invalidVertex) {
        const CSR::Vertex* transitVertex = csr->get_vertices() + transit;
        int tid = threadIdx.x % threadsToLoadTransit;
        int data = ((const int*)transitVertex)[tid];
        *(((int*)&shMem.transitVertices[transitI * sizeof(CSR::Vertex)]) + tid) = data;
      }
    }
    __syncwarp();
    
    for (int transitI = 0; transitI < TRANSITS_PER_THREAD; transitI++) {
      if (TRANSITS_PER_THREAD * (fullBlockIdx) + transitI >= gridKernelTBPositionsNum) {
        continue;
      }
      __syncthreads();
      VertexID_t transit = shMem.subWarpTransits[transitI][threadIdx.x/subWarpSize];
      CSR::Vertex* shMemTransitVertex = ((CSR::Vertex*)(&shMem.transitVertices[transitI * sizeof(CSR::Vertex)]));
      EdgePos_t numEdgesInShMem = shMemTransitVertex->num_edges();
      const CSR::Edge* glTransitEdges = (CSR::Edge*)csr->get_edges() + shMemTransitVertex->get_start_edge_idx();
      const float* glTransitEdgeWeights = (float*)(CSR::Edge*)csr->get_weights() + shMemTransitVertex->get_start_edge_idx();
      float maxWeight = shMemTransitVertex->get_max_weight();

      if (threadIdx.x == 0) {
        shMem.invalidateCache = shMem.transitForTB != transit || transitI == 0;
        shMem.transitForTB = transit;
      }

      __syncthreads();
      if (CACHE_EDGES && shMem.invalidateCache) {
        for (int i = threadIdx.x; i < min(CACHE_SIZE, numEdgesInShMem); i += blockDim.x) {
          if (ONDEMAND_CACHING) {
            if (i < STATIC_CACHE_SIZE)
              edgesInShMem[i] = glTransitEdges[i];
            else 
              edgesInShMem[i] = -1;
          } else {
            edgesInShMem[i] = glTransitEdges[i];
          }
        }
      }
  
      if (CACHE_WEIGHTS && shMem.invalidateCache) {
        for (int i = threadIdx.x; i < min(CACHE_SIZE, numEdgesInShMem); i += blockDim.x) {
          edgeWeightsInShMem[i] = (ONDEMAND_CACHING) ? -1 : glTransitEdgeWeights[i];
        }
      }

      __syncthreads();
      VertexID_t transitNeighborIdx = threadIdx.x % subWarpSize;
      bool continueExecution = true;
      continueExecution = (transitNeighborIdx < stepSize); 
      bool threadValid = continueExecution && (transit == shMem.transitForTB);
      VertexID_t neighbor = invalidVertex;
      VertexID_t sampleIdx;
      transitIdx = shMem.mapStartPos[transitI] + threadIdx.x/subWarpSize; //threadId/stepSize(step);
      sampleIdx = shMem.subWarpSampleIdx[transitI][threadIdx.x/subWarpSize];

      if (threadValid) {        
        //TODO: Set this based on the input template parameters.
        typedef CachedArray<CSR::Edge, CACHE_SIZE, ONDEMAND_CACHING, STATIC_CACHE_SIZE> CachedEdges;
        typedef CachedArray<float, CACHE_SIZE, ONDEMAND_CACHING, STATIC_CACHE_SIZE> CachedWeights;

        CachedEdges cachedEdges = {glTransitEdges, edgesInShMem};
        CachedWeights cachedWeights = {glTransitEdgeWeights, edgeWeightsInShMem};

        if (numEdgesInShMem > 0)
          neighbor = App().template next<SampleType, CachedEdges, CachedWeights>(step, csr, &transit, sampleIdx, &samples[sampleIdx-deviceFirstSample], maxWeight, 
                                                                cachedEdges, cachedWeights,
                                                                numEdgesInShMem, transitNeighborIdx, &localRandState);        
        if (step != App().steps() - 1) {
          //No need to store at last step
          samplesToTransitKeys[transitIdx] = sampleIdx; //TODO: Update this for khop to transitIdx + transitNeighborIdx
          if (App().hasExplicitTransits()) {
            VertexID_t newTransit = App().stepTransits(step, sampleIdx, samples[sampleIdx-deviceFirstSample], transitIdx, &localRandState);
            samplesToTransitValues[transitIdx] = newTransit != -1 ? newTransit : invalidVertex;
          } else {
            samplesToTransitValues[transitIdx] = neighbor != -1 ? neighbor : invalidVertex;
          }
        }
      }

      __syncwarp();
      {
        EdgePos_t insertionPos = transitNeighborIdx; 
        if (numTransitsPerStepForSample > 1) {
          if (step == 0) {
            insertionPos = transitNeighborIdx;
          } else {
            EdgePos_t insertionStartPosForTransit = 0;
            //FIXME: 
            if (threadValid && continueExecution && isValidSampledVertex(neighbor, invalidVertex) && threadIdx.x % subWarpSize == 0) {
              insertionStartPosForTransit = utils::atomicAdd(&sampleInsertionPositions[sampleIdx-deviceFirstSample], stepSize);
            }
            insertionStartPosForTransit = __shfl_sync(FULL_WARP_MASK, insertionStartPosForTransit, 0, subWarpSize);
            insertionPos = finalSampleSizeTillPreviousStep + insertionStartPosForTransit + transitNeighborIdx;
          }
        } else {
          insertionPos = step;
        }

        if (continueExecution && threadValid) {
          assert(insertionPos < finalSampleSize);
        }

        if (threadValid) {
          if (combineTwoSampleStores && numTransitsPerStepForSample == 1) {
            if (step % 2 == 1) {
              int2 *ptr = (int2*)&finalSamples[(sampleIdx-deviceFirstSample)*finalSampleSize + insertionPos - 1];
              int2 res;
              res.x = transit;
              res.y = neighbor;
              *ptr = res;
              //finalSamples[sample*finalSampleSize + insertionPos] = neighbor;
            } else if (step == App().steps() - 1) {
              finalSamples[(sampleIdx-deviceFirstSample)*finalSampleSize + insertionPos] = neighbor;
            }
          } else {
            if (isValidSampledVertex(neighbor, invalidVertex))
              finalSamples[(sampleIdx-deviceFirstSample)*finalSampleSize + insertionPos] = neighbor;
          }
          //TODO: We do not need atomic instead store indices of transit in another array,
          //which can be accessed based on sample and transitIdx.
        }
      }
    }
  }
}

template<typename App>
__global__ void collectiveNeighbrsSize(const int step, GPUCSRPartition graph, 
                                       const VertexID_t invalidVertex,
                                       VertexID_t* initialSamples, 
                                       VertexID_t* finalSamples, 
                                       const size_t finalSampleSize, 
                                       EdgePos_t* sampleNeighborhoodPos,
                                       EdgePos_t* sumNeighborhoodSizes)
{
  //Assign one thread block to a sample
  __shared__ EdgePos_t neighborhoodSize;

  if (threadIdx.x == 0) {
    neighborhoodSize = 0;
  }

  __syncthreads();

  CSRPartition* csr = (CSRPartition*)&csrPartitionBuff[0];  
  VertexID_t sampleIdx = blockIdx.x;
  EdgePos_t numTransits = App().initialSampleSize(nullptr);
  //EdgePos_t numTransitsInPrevStep = numberOfTransits(step - 1);

  //TODO: Assuming step is 0
  for (int transitIdx = threadIdx.x; transitIdx < numTransits; transitIdx += blockDim.x) {
    VertexID_t transit;
    if (step == 0) 
      transit = initialSamples[sampleIdx*App().initialSampleSize(nullptr) + transitIdx];
    else 
      transit = finalSamples[sampleIdx*App().initialSampleSize(nullptr) + transitIdx];
    if (transit != invalidVertex) {
      ::atomicAdd(&neighborhoodSize, csr->get_n_edges_for_vertex(transit)); 
    }
  }

  __syncthreads();

  if (threadIdx.x == 0) {
    sampleNeighborhoodPos[sampleIdx] = ::atomicAdd(sumNeighborhoodSizes, neighborhoodSize);
  }
}

template<typename App>
__global__ void collectiveNeighborhood(const int step, GPUCSRPartition graph, 
                                       const VertexID_t invalidVertex,
                                       VertexID_t* initialSamples, 
                                       VertexID_t* finalSamples, 
                                       const size_t finalSampleSize, 
                                       EdgePos_t* sampleNeighborhoodCSRRows,
                                       VertexID_t* sampleNeighborhoodCSRCols,
                                       EdgePos_t* sampleNeighborhoodPos,
                                       EdgePos_t* sumNeighborhoodSizes)
{
  //Assign one thread block to a sample
  EdgePos_t insertionPos = 0;
  CSRPartition* csr = (CSRPartition*)&csrPartitionBuff[0];  
  VertexID_t sampleIdx = blockIdx.x;
  EdgePos_t numTransits = App().initialSampleSize(nullptr);
  //EdgePos_t numTransitsInPrevStep = numberOfTransits(step - 1);

  //TODO: Assuming step is 0
  //Copy edges from graph, vertex by vertex
  for (int transitIdx = 0; transitIdx < numTransits; transitIdx++) {
    VertexID_t transit = initialSamples[sampleIdx*App().initialSampleSize(nullptr) + transitIdx];
    EdgePos_t nEdges = csr->get_n_edges_for_vertex(transit);
    const CSR::Edge* edges = csr->get_edges(transit);
    
    sampleNeighborhoodCSRRows[sampleIdx*App().initialSampleSize(nullptr) + transitIdx] = insertionPos;

    for (int e = threadIdx.x; e < nEdges; e += blockDim.x) {
      EdgePos_t pos = sampleNeighborhoodPos[sampleIdx] + insertionPos + e;
      sampleNeighborhoodCSRCols[pos] = edges[e];
    }

    insertionPos += nEdges;
    __syncthreads();
  }
}

template<class SampleType, typename App, bool StoreAsMap>
__global__ void explicitTransitsKernel(const int step, GPUCSRPartition graph, 
                                     const VertexID_t invalidVertex,
                                     const size_t threadsExecuted, 
                                     const size_t currExecutionThreads,
                                     const size_t totalThreads,
                                     SampleType* samples,
                                     const size_t NumSamples,
                                     VertexID_t* samplesToTransitKeys,
                                     VertexID_t* samplesToTransitValues,
                                     hiprandState* randStates)
{
  //Number of threads executed are: Num of Samples * Number of Transits
  int threadId = threadIdx.x + blockDim.x * blockIdx.x;
  //__shared__ VertexID newNeigbhors[N_THREADS];
  if (threadId >= currExecutionThreads)
    return;
  
  hiprandState* randState = &randStates[threadId];
  threadId += threadsExecuted;
  if (threadId >= totalThreads)
    return;
  EdgePos_t numTransits = numberOfTransits<App>(step - 1);
  EdgePos_t sampleIdx = threadId/numTransits;
  if (sampleIdx > NumSamples)
    return;

  EdgePos_t transitIdx = threadId % numTransits;
  if (App().samplingType() == CollectiveNeighborhood) {
    assert(!App().hasExplicitTransits());
  } else {
    VertexID_t transit = App().stepTransits(step, sampleIdx, samples[sampleIdx], transitIdx, randState);
    samplesToTransitValues[threadId] = transit;

    if (StoreAsMap) {
      samplesToTransitKeys[threadId] = sampleIdx;
    }
  }
}

/**
  sampleParallelKernel()    - Sample Parallel Kernel
  Arguments:
  @SampleType               : class of Sample
  @App                      : App class
  @THREADS                  : Number of threads in a thread block
  @step                     : Current executing step
  @graph                    : CSR Partition stored in GPU 
  @invalidVertex            : Value of invalid vertex
  @threadsExecuted          : Number of threads already executed
  @currExecutionThreads     : Number of grid threads in current execution
  @totalThreads             : Total number of threads this kernel will be invoked with
  @initialSamples           : Array of initial contents of all samples
  @samples                  : Array of all samples
  @NumSamples               : Number of samples
  @finalSamples             : Sampled vertices for all samples
  @finalSampleSize          : Final number of vertices in sample 
  @explicitTransits         : Array of explicit transits
  @sampleInsertionPositions : Insertion Position for sampled vertex in a Sample
  @randStates               : hiprand states

  Sample Parallel Kernel doing sampling on GPU using a sample parallel paradigm. 
  */
  template<class SampleType, typename App, int THREADS, bool WriteSampleToTransitMap>
  __global__ void sampleParallelKernel(const int step, GPUCSRPartition graph, 
                                       const size_t deviceFirstSample,
                                       const VertexID_t invalidVertex,
                                       const size_t totalThreads,
                                       VertexID_t* initialSamples,
                                       SampleType* samples,
                                       const size_t NumSamples,
                                       VertexID_t* finalSamples,
                                       const size_t finalSampleSize, 
                                       VertexID_t* samplesToTransitMapKeys, 
                                       VertexID_t* samplesToTransitMapValues,
                                       EdgePos_t* sampleInsertionPositions,
                                       hiprandState* randStates)
  {
    __shared__ unsigned char shMemCuRand[sizeof(hiprandState)*THREADS];
  
    int threadId = threadIdx.x + blockDim.x * blockIdx.x;
  
    graph.device_csr = (CSRPartition*)&csrPartitionBuff[0];
    
    hiprandState* curandSrcPtr;
    bool COALESCE_CURAND_LOAD = true;
    if (COALESCE_CURAND_LOAD) {
      //Load hiprand states efficiently in registers
      const int intsInRandState = sizeof(hiprandState)/sizeof(int);
      int* shStateBuff = (int*)&shMemCuRand[0];
  
      int* randStatesAsInts = (int*)randStates;
    
      for (int i = threadIdx.x; i < intsInRandState*blockDim.x; i += blockDim.x) {
        shStateBuff[i] = randStatesAsInts[i + blockDim.x*blockIdx.x];
      }
  
      __syncthreads();
      curandSrcPtr = (hiprandState*)(&shStateBuff[threadIdx.x*intsInRandState]);
    } else {
      curandSrcPtr = &randStates[threadId];
    }
  
    hiprandState localRandState = *curandSrcPtr;
  
    for (; threadId < totalThreads; threadId += gridDim.x * blockDim.x) {
      if (threadId >= totalThreads)
        return;
  
      EdgePos_t numTransits = numberOfTransits<App>(step);
      EdgePos_t numTransitsInPrevStep = numberOfTransits<App>(step - 1);
      VertexID_t sampleIdx = threadId / numTransits + deviceFirstSample; //(threadId / numTransits) ranges from [0, # of samples for this device]
  
      VertexID_t* transits = nullptr;
      VertexID_t singleTransit = 0;
      EdgePos_t numTransitsInNeghbrhood = 0;
      //TODO: Template this kernel based on the sampling type
      if (App().samplingType() == CollectiveNeighborhood) {
        assert(!App().hasExplicitTransits());
        numTransitsInNeghbrhood = numberOfTransits<App>(step);
        if (step == 0) {
          transits = &initialSamples[(sampleIdx - deviceFirstSample)*App().initialSampleSize(nullptr)];
        } else {
          size_t verticesAddTillPreviousStep = stepSizeAtStep<App>(step - 2);
  
          transits = &finalSamples[(sampleIdx - deviceFirstSample)*finalSampleSize + verticesAddTillPreviousStep];
        }
      } else {
        if (step == 0) {
          EdgePos_t transitIdx = threadId % App().initialSampleSize(nullptr);
          singleTransit = initialSamples[(sampleIdx - deviceFirstSample)*App().initialSampleSize(nullptr) + transitIdx];
        } else if (App().hasExplicitTransits()) {
          singleTransit = samplesToTransitMapValues[(sampleIdx - deviceFirstSample)*numTransitsInPrevStep + (threadId % numTransits) / numTransitsInPrevStep];
        } else {
          singleTransit = finalSamples[(sampleIdx - deviceFirstSample)*finalSampleSize + (step - 1) * numTransits + (threadId % numTransits) % numTransitsInPrevStep];
        }
  
        numTransitsInNeghbrhood = 1;
        transits = &singleTransit;
      }
      
      VertexID_t neighbor = invalidVertex;
      VertexID_t neighbrID = threadId % App().stepSize(step) ;//(threadId % numTransits) % numTransitsInPrevStep;
      VertexID_t transitID = (threadId % numTransits) / App().stepSize(step);

      if (*transits != invalidVertex and *transits != -1) {
        EdgePos_t numTransitEdges = 0;
  
        for (int i = 0; i < numTransitsInNeghbrhood; i++) {
          if (!graph.device_csr->has_vertex(transits[i])) {
            // printf("transits[%d] %d step %d \n", i, transits[i], step);
            return;
          }
          numTransitEdges += graph.device_csr->get_n_edges_for_vertex(transits[i]);
        }
        
        const CSR::Edge* transitEdges = (App().samplingType() == CollectiveNeighborhood) ? nullptr : graph.device_csr->get_edges(*transits);
        const float* transitEdgeWeights = (App().samplingType() == CollectiveNeighborhood) ? nullptr : graph.device_csr->get_weights(*transits);
        const float maxWeight = (App().samplingType() == CollectiveNeighborhood) ? 0.0 : graph.device_csr->get_max_weight(*transits);

        neighbor = App().template next<SampleType, const CSR::Edge*, const float*> (step, graph.device_csr, transits, sampleIdx, &samples[(sampleIdx - deviceFirstSample)], maxWeight, 
          transitEdges, transitEdgeWeights, numTransitEdges, neighbrID, &localRandState);
        if (neighbor == -1) {
          neighbor = invalidVertex;
        }
    #if 0
        //search if neighbor has already been selected.
        //we can do that in register if required
        newNeigbhors[threadIdx.x] = neighbor;

        bool found = false;
        for (int i = 0; i < N_THREADS; i++) {
          if (newNeigbhors[i] == neighbor) {
            found = true;
            // break;
          }
        }

        __syncwarp();
        if (found) {
          neighbor = next(step, transit, sample, transitEdges, numTransitEdges, 
            transitNeighborIdx, randState);;
        }
    #endif
      }
  
      if (WriteSampleToTransitMap) {
        samplesToTransitMapKeys[threadId] = sampleIdx;
        samplesToTransitMapValues[threadId] = neighbor;
      }

      EdgePos_t insertionPos = 0; 
  
      size_t finalSampleSizeTillPreviousStep = 0;
      size_t neighborsToSampleAtStep = 1;
      for (int _s = 0; _s < step; _s++) {
        neighborsToSampleAtStep *= App().stepSize(_s);
        finalSampleSizeTillPreviousStep += neighborsToSampleAtStep;
      }

      if (App().outputFormat() == AdjacencyMatrix && App().samplingType() == CollectiveNeighborhood) {
        assert (neighbor != invalidVertex);
        finalSamples[sampleIdx*finalSampleSize + stepSizeAtStep<App>(step - 1) + neighbrID] = neighbor;
      } else if (App().outputFormat() == SampledVertices && App().samplingType() == IndividualNeighborhood) {
        if (numberOfTransits<App>(step) > 1) {    
          insertionPos = utils::atomicAdd(&sampleInsertionPositions[sampleIdx], 1);//finalSampleSizeTillPreviousStep + (threadId % numTransits);//
        } else {
          insertionPos = step;
        }
        
        finalSamples[(sampleIdx - deviceFirstSample)*finalSampleSize + insertionPos] = neighbor;
      }
  
      //TODO: We do not need atomic instead store indices of transit in another array,
      //which can be accessed based on sample and transitIdx.
    }
  
    //Write back the updated hiprand states
    if (COALESCE_CURAND_LOAD) {
      const int intsInRandState = sizeof(hiprandState)/sizeof(int);
      hiprandState* shStateBuff = (hiprandState*)&shMemCuRand[0];
      shStateBuff[threadIdx.x] = localRandState;
      __syncthreads();
  
      int* shStateBuffAsInts = (int*)&shStateBuff[0];
      int* randStatesAsInts = (int*)randStates;
    
      for (int i = threadIdx.x; i < intsInRandState*blockDim.x; i += blockDim.x) {
        randStatesAsInts[i + blockDim.x*blockIdx.x] = shStateBuffAsInts[i];
      }
    } else {
      *curandSrcPtr = localRandState;
    }
  }

template<typename App, int TB_THREADS, TransitKernelTypes kTy, bool WRITE_KERNELTYPES>
__global__ void partitionTransitsInKernels(int step, EdgePos_t* uniqueTransits, EdgePos_t* uniqueTransitCounts, 
                                           EdgePos_t* transitPositions,
                                           EdgePos_t uniqueTransitCountsNum, VertexID_t invalidVertex,
                                           EdgePos_t* gridKernelTransits, EdgePos_t* gridKernelTransitsNum,
                                           EdgePos_t* threadBlockKernelTransits, EdgePos_t* threadBlockKernelTransitsNum,
                                           EdgePos_t* subWarpKernelTransits, EdgePos_t* subWarpKernelTransitsNum,
                                           EdgePos_t* identityKernelTransits, EdgePos_t* identityKernelTransitsNum,
                                           int* kernelTypeForTransit, VertexID_t* transitToSamplesKeys) 
{
  //__shared__ EdgePos_t insertionPosOfThread[TB_THREADS];
  const int SHMEM_SIZE = 7*TB_THREADS;
  // __shared__ EdgePos_t trThreadBlocks[TB_THREADS];
  // __shared__ EdgePos_t trStartPos[TB_THREADS];
  typedef hipcub::BlockScan<int, TB_THREADS> BlockScan;
  __shared__ typename BlockScan::TempStorage temp_storage;
  //__shared__ typename BlockScan::TempStorage temp_storage2;
  __shared__ EdgePos_t shGridKernelTransits[SHMEM_SIZE];
  //__shared__ EdgePos_t warpsLastThreadVals;
  __shared__ EdgePos_t threadToTransitPrefixSum[TB_THREADS];
  __shared__ EdgePos_t threadToTransitPos[TB_THREADS];
  __shared__ VertexID_t threadToTransit[TB_THREADS];
  __shared__ EdgePos_t totalThreadGroups;
  __shared__ EdgePos_t threadGroupsInsertionPos;
//  __shared__ EdgePos_t gridKernelTransitsIter;

  int threadId = blockIdx.x * blockDim.x + threadIdx.x;

  if (threadIdx.x == 0) {
    totalThreadGroups = 0;
  //  gridKernelTransitsIter = 0;
  }

  for (int i = threadIdx.x; i < SHMEM_SIZE; i += blockDim.x) {
    shGridKernelTransits[i] = 0;
  }

  __syncthreads();
  
  VertexID_t transit = uniqueTransits[threadId];
  EdgePos_t trCount = (threadId >= uniqueTransitCountsNum || transit == invalidVertex) ? -1: uniqueTransitCounts[threadId];
  EdgePos_t trPos = (threadId >= uniqueTransitCountsNum || transit == invalidVertex) ? -1: transitPositions[threadId];
  int subWarpSize = subWarpSizeAtStep<App>(step);
  
  int kernelType = -1;
  if (useGridKernel && trCount * subWarpSize >= LoadBalancing::LoadBalancingThreshold::GridLevel) {    
    kernelType = TransitKernelTypes::GridKernel;
  } else if (useThreadBlockKernel && trCount * subWarpSize >= LoadBalancing::LoadBalancingThreshold::BlockLevel) {
    kernelType = TransitKernelTypes::ThreadBlockKernel;
  } else if (useSubWarpKernel && trCount * subWarpSize >= LoadBalancing::LoadBalancingThreshold::SubWarpLevel) {
    kernelType = TransitKernelTypes::SubWarpKernel;
  } else {
    kernelType = TransitKernelTypes::IdentityKernel;
  }
  
  if (WRITE_KERNELTYPES) {
    if (threadId < uniqueTransitCountsNum && kernelType != IdentityKernel && transit != invalidVertex) {
      kernelTypeForTransit[transit] = kernelType;
    } 

    if (kernelType == IdentityKernel && transit != invalidVertex && trCount !=-1) {
      *identityKernelTransitsNum = 1;
    }
  }

  __syncthreads();

  // if (kernelType == TransitKernelTypes::ThreadBlockKernel && WRITE_KERNELTYPES && transit < 20000) {
  //   printf("1769: transit %d trCount %d trPos %d\n", transit, trCount, trPos);
  // }
  //TODO: Remove unnecessary "__syncthreads();" statements

  //for (int kTy = 1; kTy < TransitKernelTypes::SubWarpKernel; kTy++)
  {
    EdgePos_t numThreadGroups = 0;
    EdgePos_t* glKernelTransitsNum, *glKernelTransits;
    const int threadGroupSize = (kTy == TransitKernelTypes::GridKernel) ? LoadBalancing::LoadBalancingThreshold::GridLevel/subWarpSize : 
                                ((kTy == TransitKernelTypes::ThreadBlockKernel) ? LoadBalancing::LoadBalancingThreshold::BlockLevel/subWarpSize : 
                                ((kTy == TransitKernelTypes::SubWarpKernel) ? LoadBalancing::LoadBalancingThreshold::SubWarpLevel : -1));

    if (kTy == TransitKernelTypes::GridKernel && useGridKernel) {
      if (kernelType == TransitKernelTypes::GridKernel) {
        numThreadGroups = DIVUP(trCount, threadGroupSize);
        threadToTransitPos[threadIdx.x] = trPos;
        threadToTransit[threadIdx.x] = transit;
      } else {
        numThreadGroups = 0;
        threadToTransitPos[threadIdx.x] = 0;
        threadToTransit[threadIdx.x] = -1;
      } 
      glKernelTransitsNum = gridKernelTransitsNum;
      glKernelTransits = gridKernelTransits;
    } else if (kTy == TransitKernelTypes::ThreadBlockKernel && useThreadBlockKernel) {
      if (kernelType == TransitKernelTypes::ThreadBlockKernel) {
        numThreadGroups = DIVUP(trCount, threadGroupSize);
        threadToTransitPos[threadIdx.x] = trPos;
        threadToTransit[threadIdx.x] = transit;
      } else {
        numThreadGroups = 0;
        threadToTransitPos[threadIdx.x] = 0;
        threadToTransit[threadIdx.x] = -1;
      }       
      glKernelTransitsNum = threadBlockKernelTransitsNum;
      glKernelTransits = threadBlockKernelTransits;
      // if (blockIdx.x == 0) {
      //   printf("threadIdx.x %d transit %d\n", threadIdx.x, transit);
      // }
    } else if (kTy == TransitKernelTypes::SubWarpKernel && useSubWarpKernel) {
      if (kernelType == TransitKernelTypes::SubWarpKernel) {
        numThreadGroups = DIVUP(trCount, threadGroupSize);
        threadToTransitPos[threadIdx.x] = trPos;
        threadToTransit[threadIdx.x] = transit;
      } else {
        numThreadGroups = 0;
        threadToTransitPos[threadIdx.x] = 0;
        threadToTransit[threadIdx.x] = -1;
      }       
      glKernelTransitsNum = subWarpKernelTransitsNum;
      glKernelTransits = subWarpKernelTransits;
    } else {
      return;
      // continue;
    }
    
    __syncthreads();
    //Get all grid kernel transits
    EdgePos_t prefixSumThreadData = 0;
    BlockScan(temp_storage).ExclusiveSum(numThreadGroups, prefixSumThreadData);
    
    __syncthreads();
    
    if (threadIdx.x == blockDim.x - 1) {
      totalThreadGroups = prefixSumThreadData + numThreadGroups;
      // if (kTy == 2 && blockIdx.x == 27) printf("totalThreadGroups %d kTy %d blockIdx.x %d\n", totalThreadGroups, kTy, blockIdx.x);
      threadGroupsInsertionPos = ::atomicAdd(glKernelTransitsNum, totalThreadGroups);
    }
    __syncthreads();

    threadToTransitPrefixSum[threadIdx.x] = prefixSumThreadData;
    
    __syncthreads();
    
    // if (totalThreadGroups != 0 and numThreadGroups != 0) {
    //   printf("threadIdx.x %d blockIdx.x %d tr %d trPos %d numThreadGroups %d totalThreadGroups %d prefixSumThreadData %d\n", threadIdx.x, blockIdx.x, transit, trPos, numThreadGroups, totalThreadGroups, prefixSumThreadData);
    // }
    
    for (int tgIter = 0; tgIter < totalThreadGroups; tgIter += SHMEM_SIZE) {
      __syncthreads();
      for (int i = threadIdx.x; i < SHMEM_SIZE; i += blockDim.x) {
        shGridKernelTransits[i] = 0;
      }
    
      __syncthreads();
      
      int prefixSumIndex = prefixSumThreadData - tgIter;
      if (prefixSumIndex < 0 && prefixSumIndex + numThreadGroups > 0) {
        prefixSumIndex = 0;
      }
      if (numThreadGroups > 0) {
        if (prefixSumIndex >= 0 && prefixSumIndex < SHMEM_SIZE) {
          shGridKernelTransits[prefixSumIndex] = threadIdx.x;
        }
      }
      
      __syncthreads();

      for (int tbs = threadIdx.x; tbs < DIVUP(min(SHMEM_SIZE, totalThreadGroups - tgIter), TB_THREADS)*TB_THREADS; tbs += blockDim.x) {
        __syncthreads();
        int d = 0, e = 0;
        if (tbs < TB_THREADS) {
          d = (tbs < totalThreadGroups) ? shGridKernelTransits[tbs] : 0;
        } else if (threadIdx.x == 0) {
          d = (tbs < totalThreadGroups) ? max(shGridKernelTransits[tbs], shGridKernelTransits[tbs-1]): 0;
        } else {
          d = (tbs < totalThreadGroups) ? shGridKernelTransits[tbs] : 0;
        }
        // if (blockIdx.x == 1 && !(d < blockDim.x && d >= 0)) {
        //   printf("d %d blockIdx.x %d tbs %d tgIter %d kTy %d tbKernelType %d totalThreadGroups %d\n", 
        //           d, blockIdx.x, tbs, tgIter, kTy, TransitKernelTypes::ThreadBlockKernel, totalThreadGroups);
        // }
        // if (threadIdx.x == 0) {
        //   for (int ii = 0; ii < sizeof(temp_storage)/sizeof(int); ii++) {
        //     ((int*)&temp_storage)[ii] = 0; 
        //   }
        // }
        __syncthreads();
        BlockScan(temp_storage).InclusiveScan(d, e, hipcub::Max());
        __syncthreads();

        if (tbs < totalThreadGroups) {
          shGridKernelTransits[tbs] = e;
        }

        // if (blockIdx.x==23 && tbs == blockDim.x - 1 && tgIter == 0 && kTy == 2) {
        //   printf("e %d blockIdx.x %d tbs %d tgIter %d kTy %d tbKernelType %d totalThreadGroups %d\n", 
        //           e, blockIdx.x, tbs, tgIter, kTy, TransitKernelTypes::ThreadBlockKernel, totalThreadGroups);
        // }

        // if (blockIdx.x==23 && !(e < blockDim.x && e >= 0)) {
        // if (blockIdx.x == 0 && tgIter == 0 && kTy == 2){//} && (tbs >= 2760 && tbs <= 2763) && tgIter == 78848) {
        //   printf("d %d e %d blockIdx.x %d tbs %d tgIter %d kTy %d tbKernelType %d totalThreadGroups %d prefixSumIndex %d\n", 
        //           d, e, blockIdx.x, tbs, tgIter, kTy, TransitKernelTypes::ThreadBlockKernel, totalThreadGroups, prefixSumIndex);
        // }
          
        __syncthreads();

        if (tbs + tgIter < totalThreadGroups) {
          EdgePos_t xx = shGridKernelTransits[tbs];
          assert(xx >= 0 && xx < blockDim.x);
          int previousTrPrefixSum = (tbs < totalThreadGroups && xx >= 0) ? threadToTransitPrefixSum[xx] : 0;

          EdgePos_t startPos = threadToTransitPos[xx];
          EdgePos_t pos = startPos + threadGroupSize*(tbs  + tgIter - previousTrPrefixSum);
          //assert(tbs >= 0 && tbs < SHMEM_SIZE);
          VertexID_t transit = threadToTransit[xx];
          if (transit != -1) {
            int idx = threadGroupsInsertionPos + tbs + tgIter;
            // if (idx < 0 || idx >= 4576926*10) {
            //   printf("idx %d\n", idx);
            // }
            glKernelTransits[idx] = pos;
            // if (blockIdx.x == 0 && !(transitToSamplesKeys[pos] == transit)) {
            //   printf("blockIdx.x %d shGridKernelTransits[tbs] %d tbs %d tgIter %d startPos %d pos %d expectedTr %d threadTr %d kernelTy %d\n", 
            //         blockIdx.x, shGridKernelTransits[tbs], tbs, tgIter, startPos, pos, transitToSamplesKeys[pos], transit, kTy);
            // }
            assert(kernelTypeForTransit[transit] == kTy);
            // if (transitToSamplesKeys[pos] != transit) {
            //   printf("blockIdx.x %d shGridKernelTransits[tbs] %d tbs %d tgIter %d startPos %d pos %d expectedTr %d threadTr %d kernelTy %d\n", blockIdx.x, shGridKernelTransits[tbs], tbs, tgIter, startPos, pos, transitToSamplesKeys[pos], transit, kTy);
            // }
            assert(transitToSamplesKeys[pos] == transit);
          }
        }

        __syncthreads();
      }

      __syncthreads();
    }

    // if (threadIdx.x==0){
    //   for (int i = 0; i < totalThreadGroups; i++) {
    //    // printf("blockIdx.x %d gridKernelTransits[%d] %d step %d\n", blockIdx.x, i, gridKernelTransits[threadGroupsInsertionPos + i], step);
    //   }
    // }

    __syncthreads();
  }

  // if (threadIdx.x+blockIdx.x*blockDim.x==0) {
  //   printf("subWarpKernelTransitsNum %d\n", *subWarpKernelTransitsNum);
  // }
  #if 0
  int done = 0;
  int startCopyingIteration = prefixSumThreadData/SHMEM_SIZE;
  int endCopyingIteration = (prefixSumThreadData + numThreadGroups)/SHMEM_SIZE;

  __syncthreads();

  for (int tbs = 0; tbs < gridTotalTBs; tbs += SHMEM_SIZE) {
    if (trPos >= 0 && numThreadBlocks > 0 && done < numThreadBlocks && tbs/SHMEM_SIZE >= startCopyingIteration && tbs/SHMEM_SIZE <= endCopyingIteration) {
      int todo;
      for (todo = 0; todo < min(numThreadBlocks-done, SHMEM_SIZE); todo++) {
        int idx = prefixSumThreadData + done - tbs + todo;
        if (idx >= SHMEM_SIZE) {
          break;
        }
        if (idx < 0 || idx >= SHMEM_SIZE) {
          printf("idx %d prefixSum %d done %d tbs %d todo %d\n", idx, prefixSumThreadData, done, tbs, todo);
        }
        shGridKernelTransits[idx] = trPos + LoadBalancing::LoadBalancingThreshold::GridLevel*(todo+done);
      }
      done += todo;
    }

    __syncthreads();

    for (EdgePos_t i = threadIdx.x; i < min(SHMEM_SIZE, gridTotalTBs - tbs); i+=blockDim.x) {
      gridKernelTransits[gridInsertionPos + tbs + i] = shGridKernelTransits[i];
    }
    __syncthreads();
  }
  #endif
}

__global__ void invalidVertexStartPos(int step, VertexID_t* transitToSamplesKeys, size_t totalTransits, 
                                      const VertexID_t invalidVertex, EdgePos_t* outputStartPos)
{
  int threadId = threadIdx.x + blockIdx.x*blockDim.x;

  if (threadId >= totalTransits) {
    return;
  }

  //If first transit is invalid.
  if (threadId == 0) {
    if (transitToSamplesKeys[0] == invalidVertex) {
      *outputStartPos = 0;
    }
    // printf("outputStartPos %d\n", *outputStartPos);
    return;
  }

  //TODO: Optimize this using overlaped tilling
  if (transitToSamplesKeys[threadId - 1] != invalidVertex && 
      transitToSamplesKeys[threadId] == invalidVertex)
  {
    *outputStartPos = threadId;
    return;
      // printf("outputStartPos %d\n", *outputStartPos);
  }

  //If no transit is invalid 
  // if (threadId == totalTransits - 1) {
  //   printf("1666: threadIdx.x %d v %d invalidVertex %d\n", threadId, transitToSamplesKeys[threadId], invalidVertex);
  //   *outputStartPos = totalTransits - 1;
  // }
}

__global__ void init_curand_states(hiprandState* states, size_t num_states)
{
  int thread_id = blockIdx.x*blockDim.x + threadIdx.x;
  if (thread_id < num_states)
    hiprand_init(thread_id, threadIdx.x, 0, &states[thread_id]);
}

CSR* loadGraph(Graph& graph, char* graph_file, char* graph_type, char* graph_format)
{
  CSR* csr;

   //Load Graph
   if (strcmp(graph_type, "adj-list") == 0) {
    if (strcmp(graph_format, "text") == 0) {
      graph.load_from_adjacency_list(graph_file);
      //Convert graph to CSR format
      csr = new CSR(graph.get_vertices().size(), graph.get_n_edges());
      csr_from_graph (csr, graph);
      return csr;
    }
    else {
      printf ("graph_format '%s' not supported for graph_type '%s'\n", 
              graph_format, graph_type);
      return nullptr;
    }
  } else if (strcmp(graph_type, "edge-list") == 0) {
    if (strcmp(graph_format, "binary") == 0) {
      graph.load_from_edge_list_binary(graph_file, true);
      csr = new CSR(graph.get_vertices().size(), graph.get_n_edges());
      csr_from_graph (csr, graph);
      return csr;
    } else if (strcmp(graph_format, "text") == 0) {
      FILE* fp = fopen (graph_file, "r");
      if (fp == nullptr) {
        std::cout << "File '" << graph_file << "' not found" << std::endl;
        return nullptr;
      }
      graph.load_from_edge_list_txt(fp, true);
      fclose (fp);
      csr = new CSR(graph.get_vertices().size(), graph.get_n_edges());
      csr_from_graph (csr, graph);
      return csr;
    } else {
      printf ("graph_format '%s' not supported for graph_type '%s'\n", 
              graph_format, graph_type);
      return nullptr;
    }
  } else {
    printf("Incorrect graph file type '%s'\n", graph_type);
    return nullptr;
  }

  return nullptr;
}

template<typename NextDoorData>
std::vector<GPUCSRPartition> transferCSRToGPUs(NextDoorData& data,  CSR* csr)
{
  //Assume that whole graph can be stored in GPU Memory.
  //Hence, only one Graph Partition is created.
  CSRPartition full_partition = CSRPartition (0, csr->get_n_vertices() - 1, 0, csr->get_n_edges() - 1, 
                                              csr->get_vertices(), csr->get_edges(), csr->get_weights());
  std::vector<GPUCSRPartition> gpuCSRPartitions;
  //Copy full graph to GPU
  for (int device = 0; device < data.devices.size(); device++) {
    GPUCSRPartition gpuCSRPartition;
    CHK_CU(hipSetDevice(data.devices[device]));
    CSRPartition deviceCSRPartition = copyPartitionToGPU(full_partition, gpuCSRPartition);
    gpuCSRPartition.device_csr = (CSRPartition*)csrPartitionBuff;
    CHK_CU(hipMemcpyToSymbol(HIP_SYMBOL(csrPartitionBuff), &deviceCSRPartition, sizeof(CSRPartition)));
    gpuCSRPartitions.push_back(gpuCSRPartition);
  }
  return gpuCSRPartitions;
}

template<typename App>
int getFinalSampleSize()
{
  size_t finalSampleSize = 0;
  size_t neighborsToSampleAtStep = 1;
  for (int step = 0; step < App().steps(); step++) {
    if (App().samplingType() == SamplingType::CollectiveNeighborhood) {
      neighborsToSampleAtStep = App().stepSize(step);
    } else {
      neighborsToSampleAtStep *= App().stepSize(step);
    }

    finalSampleSize += neighborsToSampleAtStep;
  }

  return finalSampleSize;
}

template<typename SampleType, typename App>
bool allocNextDoorDataOnGPU(CSR* csr, NextDoorData<SampleType, App>& data)
{
  char* deviceList;
  if ((deviceList = getenv("CUDA_DEVICES")) != nullptr) {
    std::string deviceListStr = deviceList;

    std::stringstream ss(deviceListStr);
    if (ss.peek() == '[')
        ss.ignore();
    for (int i; ss >> i;) {
      data.devices.push_back(i);    
      if (ss.peek() == ',')
        ss.ignore();
    }
    if (ss.peek() == ']')
      ss.ignore();
  } else {
    data.devices = {0};
  }

  std::cout << "Using GPUs: [";
  for (auto d : data.devices) {
    std::cout << d << ",";
  }
  std::cout << "]" << std::endl;

  int maxV = 0;
  // printf("App().numSamples(csr) %d\n", App().numSamples(csr));
  for (int sampleIdx = 0; sampleIdx < App().numSamples(csr); sampleIdx++) {
    SampleType sample = App().template initializeSample<SampleType>(csr, sampleIdx);
    data.samples.push_back(sample);
    auto initialVertices = App().initialSample(sampleIdx, csr, data.samples[data.samples.size() - 1]);
    if ((EdgePos_t)initialVertices.size() != App().initialSampleSize(csr)) {
      //We require that number of vertices in sample initially are equal to the initialSampleSize
      printf ("initialSampleSize '%d' != initialSample(%d).size() '%ld'\n", 
              App().initialSampleSize(csr), sampleIdx, initialVertices.size());
      abort();
    }

    data.initialContents.insert(data.initialContents.end(), initialVertices.begin(), initialVertices.end());
    for (auto v : initialVertices)
      data.initialTransitToSampleValues.push_back(sampleIdx);
  }

  for (auto vertex : csr->iterate_vertices()) {
    maxV = (maxV < vertex) ? vertex : maxV;
  }
  //Size of each sample output
  size_t maxNeighborsToSample = App().initialSampleSize(csr); //TODO: Set initial vertices
  for (int step = 0; step < App().steps() - 1; step++) {
    if (App().samplingType() == SamplingType::CollectiveNeighborhood) {
      maxNeighborsToSample = max((long)App().stepSize(step), maxNeighborsToSample);
    } else {
      maxNeighborsToSample *= App().stepSize(step);
    }
  }

  int finalSampleSize = getFinalSampleSize<App>();
  std::cout << "Final Size of each sample: " << finalSampleSize << std::endl;
  std::cout << "Maximum Neighbors Sampled at each step: " << maxNeighborsToSample << std::endl;
  std::cout << "Number of Samples: " << App().numSamples(csr) << std::endl;
  data.INVALID_VERTEX = csr->get_n_vertices();
  int maxBits = 0;
  while ((data.INVALID_VERTEX >> maxBits) != 0) {
    maxBits++;
  }
  
  data.maxBits = maxBits;
  
  // size_t free = 0, total = 0;
  // CHK_CU(hipMemGetInfo(&free, &total));
  // printf("free memory %ld nextDoorData.samples.size() %ld maxNeighborsToSample %ld\n", free, data.samples.size(), maxNeighborsToSample);
  const size_t numSamples = data.samples.size();
  //Allocate storage for final samples on GPU
  data.hFinalSamples = std::vector<VertexID_t>(finalSampleSize*numSamples);
  data.dSamplesToTransitMapKeys = std::vector<VertexID_t*>(data.devices.size(), nullptr);
  data.dSamplesToTransitMapValues = std::vector<VertexID_t*>(data.devices.size(), nullptr);
  data.dTransitToSampleMapKeys = std::vector<VertexID_t*>(data.devices.size(), nullptr);
  data.dTransitToSampleMapValues = std::vector<VertexID_t*>(data.devices.size(), nullptr);
  data.dSampleInsertionPositions = std::vector<VertexID_t*>(data.devices.size(), nullptr);
  data.dNeighborhoodSizes = std::vector<EdgePos_t*>(data.devices.size(), nullptr);
  data.dCurandStates = std::vector<hiprandState*>(data.devices.size(), nullptr);
  data.maxThreadsPerKernel = std::vector<size_t>(data.devices.size(), 0);
  data.dFinalSamples = std::vector<VertexID_t*>(data.devices.size(), nullptr);
  data.dInitialSamples = std::vector<VertexID_t*>(data.devices.size(), nullptr);
  data.dOutputSamples = std::vector<SampleType*>(data.devices.size(), nullptr);
  const size_t numDevices = data.devices.size();
  for(auto deviceIdx = 0; deviceIdx < data.devices.size(); deviceIdx++) {
    auto device = data.devices[deviceIdx];
    //Per Device Allocation
    CHK_CU(hipSetDevice(device));
    
    const size_t perDeviceNumSamples = PartDivisionSize(numSamples, deviceIdx, numDevices);
    const size_t deviceSampleStartPtr = PartStartPointer(numSamples, deviceIdx, numDevices);

    //Allocate storage and copy initial samples on GPU
    size_t partDivisionSize = App().initialSampleSize(csr)*perDeviceNumSamples;
    size_t partStartPtr = App().initialSampleSize(csr)*deviceSampleStartPtr;
    CHK_CU(hipMalloc(&data.dInitialSamples[deviceIdx], sizeof(VertexID_t)*partDivisionSize));
    CHK_CU(hipMemcpy(data.dInitialSamples[deviceIdx], &data.initialContents[0] + partStartPtr, 
                      sizeof(VertexID_t)*partDivisionSize, hipMemcpyHostToDevice));

    //Allocate storage for samples on GPU
    if (sizeof(SampleType) > 0) {
      CHK_CU(hipMalloc(&data.dOutputSamples[deviceIdx], sizeof(SampleType)*perDeviceNumSamples));
      CHK_CU(hipMemcpy(data.dOutputSamples[deviceIdx], &data.samples[0] + deviceSampleStartPtr, sizeof(SampleType)*perDeviceNumSamples, 
                        hipMemcpyHostToDevice));
    }

    CHK_CU(hipMalloc(&data.dFinalSamples[deviceIdx], sizeof(VertexID_t)*finalSampleSize*perDeviceNumSamples));
    gpu_memset(data.dFinalSamples[deviceIdx], data.INVALID_VERTEX, finalSampleSize*perDeviceNumSamples);
    
    //Samples to Transit Map
    CHK_CU(hipMalloc(&data.dSamplesToTransitMapKeys[deviceIdx], sizeof(VertexID_t)*perDeviceNumSamples*maxNeighborsToSample));
    CHK_CU(hipMalloc(&data.dSamplesToTransitMapValues[deviceIdx], sizeof(VertexID_t)*perDeviceNumSamples*maxNeighborsToSample));

    //Transit to Samples Map
    CHK_CU(hipMalloc(&data.dTransitToSampleMapKeys[deviceIdx], sizeof(VertexID_t)*perDeviceNumSamples*maxNeighborsToSample));
    CHK_CU(hipMalloc(&data.dTransitToSampleMapValues[deviceIdx], sizeof(VertexID_t)*perDeviceNumSamples*maxNeighborsToSample));

    //Same as initial values of samples for first iteration
    CHK_CU(hipMemcpy(data.dTransitToSampleMapKeys[deviceIdx], &data.initialContents[0] + partStartPtr, sizeof(VertexID_t)*partDivisionSize, 
                      hipMemcpyHostToDevice));
    CHK_CU(hipMemcpy(data.dTransitToSampleMapValues[deviceIdx], &data.initialTransitToSampleValues[0] + partStartPtr, 
                      sizeof(VertexID_t)*partDivisionSize, hipMemcpyHostToDevice));
    //Insertion positions per transit vertex for each sample
    CHK_CU(hipMalloc(&data.dSampleInsertionPositions[deviceIdx], sizeof(EdgePos_t)*perDeviceNumSamples));

    size_t curandDataSize = maxNeighborsToSample*perDeviceNumSamples*sizeof(hiprandState);
    const size_t curandSizeLimit = 5L*1024L*1024L*sizeof(hiprandState);
    if (curandDataSize < curandSizeLimit) {
      int maxSubWarpSize = 0;
      for (int s = 0; s < App().steps(); s++) {
        maxSubWarpSize = max(maxSubWarpSize, subWarpSizeAtStep<App>(s));
      }
      //Maximum threads for a kernel should ensure that for a transit for a sample all needed
      //neighbors are sampled.
      assert(maxSubWarpSize != 0);
      data.maxThreadsPerKernel[deviceIdx] = ROUNDUP(maxNeighborsToSample*perDeviceNumSamples, maxSubWarpSize*N_THREADS);
      curandDataSize = data.maxThreadsPerKernel[deviceIdx] * sizeof(hiprandState);
    } else {
      data.maxThreadsPerKernel[deviceIdx] = curandSizeLimit/sizeof(hiprandState);
      curandDataSize = curandSizeLimit;
    }
    printf("Maximum Threads Per Kernel: %ld\n", data.maxThreadsPerKernel[deviceIdx]);
    CHK_CU(hipMalloc(&data.dCurandStates[deviceIdx], curandDataSize));
    init_curand_states<<<thread_block_size(data.maxThreadsPerKernel[deviceIdx], 256UL), 256UL>>> (data.dCurandStates[deviceIdx], data.maxThreadsPerKernel[deviceIdx]);
    CHK_CU(hipDeviceSynchronize());
    if (App().samplingType() == SamplingType::CollectiveNeighborhood) {
      CHK_CU(hipMalloc(&data.dNeighborhoodSizes[deviceIdx], sizeof(EdgePos_t)*perDeviceNumSamples));
    }
  }

  return true;
}

template<class SampleType, typename App>
void freeDeviceData(NextDoorData<SampleType, App>& data) 
{
  for(auto deviceIdx = 0; deviceIdx < data.devices.size(); deviceIdx++) {
    auto device = data.devices[deviceIdx];
    CHK_CU(hipSetDevice(device));
    CHK_CU(hipFree(data.dSamplesToTransitMapKeys[deviceIdx]));
    CHK_CU(hipFree(data.dSamplesToTransitMapValues[deviceIdx]));
    CHK_CU(hipFree(data.dTransitToSampleMapKeys[deviceIdx]));
    CHK_CU(hipFree(data.dTransitToSampleMapValues[deviceIdx]));
    CHK_CU(hipFree(data.dSampleInsertionPositions[deviceIdx]));
    CHK_CU(hipFree(data.dCurandStates[deviceIdx]));
    CHK_CU(hipFree(data.dFinalSamples[deviceIdx]));
    if (App().samplingType() == SamplingType::CollectiveNeighborhood) {
      CHK_CU(hipFree(data.dNeighborhoodSizes[deviceIdx]));
    }
    CHK_CU(hipFree(data.dInitialSamples[deviceIdx]));
    if (sizeof(SampleType) > 0) CHK_CU(hipFree(data.dOutputSamples[deviceIdx]));
  }

  //TODO:
  for (int device = 0; device < data.devices.size(); device++) {
    CHK_CU(hipSetDevice(data.devices[device]));
    CHK_CU(hipFree(data.gpuCSRPartitions[device].device_vertex_array));
    CHK_CU(hipFree(data.gpuCSRPartitions[device].device_edge_array));
    CHK_CU(hipFree(data.gpuCSRPartitions[device].device_weights_array));
  }
}

template<typename App>
void printKernelTypes(int step, CSR* csr, VertexID_t* dUniqueTransits, VertexID_t* dUniqueTransitsCounts, EdgePos_t* dUniqueTransitsNumRuns)
{
  EdgePos_t* hUniqueTransitsNumRuns = GPUUtils::copyDeviceMemToHostMem(dUniqueTransitsNumRuns, 1);
  VertexID_t* hUniqueTransits = GPUUtils::copyDeviceMemToHostMem(dUniqueTransits, *hUniqueTransitsNumRuns);
  VertexID_t* hUniqueTransitsCounts = GPUUtils::copyDeviceMemToHostMem(dUniqueTransitsCounts, *hUniqueTransitsNumRuns);

  size_t identityKernelTransits = 0, identityKernelSamples = 0, maxEdgesOfIdentityTransits = 0;
  size_t subWarpLevelTransits = 0, subWarpLevelSamples = 0, maxEdgesOfSubWarpTransits = 0, subWarpTransitsWithEdgesLessThan384 = 0, subWarpTransitsWithEdgesMoreThan384 = 0, numSubWarps = 0;
  size_t threadBlockLevelTransits = 0, threadBlockLevelSamples = 0, tbVerticesWithEdgesLessThan3K = 0, tbVerticesWithEdgesMoreThan3K = 0;
  size_t gridLevelTransits = 0, gridLevelSamples = 0, gridVerticesWithEdgesLessThan3K = 0, gridVerticesWithEdgesMoreThan3K = 0,
  gridVerticesWithEdgesLessThan1K = 0, gridVerticesWithEdgesLessThan2K = 0;
  EdgePos_t maxEdgesOfGridTransits = 0;
  int subWarpSize =  subWarpSizeAtStep<App>(step);

  for (EdgePos_t tr = 0; tr < *hUniqueTransitsNumRuns; tr++) {
    // if (tr == 0) {printf("%s:%d hUniqueTransitsCounts[0] is %d\n", __FILE__, __LINE__, hUniqueTransitsCounts[tr]);}
    if (hUniqueTransitsCounts[tr] * subWarpSize < 8) {
      identityKernelTransits++;
      identityKernelSamples += hUniqueTransitsCounts[tr];
      maxEdgesOfIdentityTransits = max(maxEdgesOfIdentityTransits, (size_t)csr->n_edges_for_vertex(tr));
    } else if (hUniqueTransitsCounts[tr] * subWarpSize <= LoadBalancing::LoadBalancingThreshold::BlockLevel && 
               hUniqueTransitsCounts[tr] * subWarpSize >= 8) {
      subWarpLevelTransits++;
      subWarpLevelSamples += hUniqueTransitsCounts[tr];
      maxEdgesOfSubWarpTransits = max(maxEdgesOfSubWarpTransits, (size_t)csr->n_edges_for_vertex(tr));
      numSubWarps += DIVUP(hUniqueTransitsCounts[tr], LoadBalancing::LoadBalancingThreshold::SubWarpLevel);
      if (csr->n_edges_for_vertex(tr) <= 96) {
        subWarpTransitsWithEdgesLessThan384 += 1;
      } else {
        subWarpTransitsWithEdgesMoreThan384 += 1;
      }
    } else if (hUniqueTransitsCounts[tr] * subWarpSize >= LoadBalancing::LoadBalancingThreshold::BlockLevel && 
               hUniqueTransitsCounts[tr] * subWarpSize < LoadBalancing::LoadBalancingThreshold::GridLevel) {
      threadBlockLevelTransits++;
      threadBlockLevelSamples += hUniqueTransitsCounts[tr];
      if (csr->n_edges_for_vertex(tr) <= 3*1024) {
        tbVerticesWithEdgesLessThan3K += 1;
      } else {
        tbVerticesWithEdgesMoreThan3K += 1;
      }
    } else {
      gridLevelTransits++;
      gridLevelSamples += hUniqueTransitsCounts[tr];
      if (csr->n_edges_for_vertex(tr) <= 3*1024) {
        if (csr->n_edges_for_vertex(tr) <= 1*1024) {
          gridVerticesWithEdgesLessThan1K += 1;
        } else if (csr->n_edges_for_vertex(tr) <= 2*1024) {
          gridVerticesWithEdgesLessThan2K += 1;
        } else 
          gridVerticesWithEdgesLessThan3K += 1;
      } else {
        gridVerticesWithEdgesMoreThan3K += 1;
      }
      maxEdgesOfGridTransits = max(maxEdgesOfGridTransits, csr->n_edges_for_vertex(tr));
    }
  }

  printf("IdentityKernelTransits: %ld, IdentityKernelSamples: %ld, MaxEdgesOfIdentityTransits: %ld\n" 
         "SubWarpLevelTransits: %ld, SubWarpLevelSamples: %ld, MaxEdgesOfSubWarpTranits: %ld, VerticesWithEdges > 384: %ld, VerticesWithEdges <= 384: %ld, NumSubWarps: %ld\n"
         "ThreadBlockLevelTransits: %ld, ThreadBlockLevelSamples: %ld, VerticesWithEdges > 3K: %ld, VerticesWithEdges < 3K: %ld\n"
         "GridLevelTransits: %ld, GridLevelSamples: %ld, VerticesWithEdges > 3K: %ld, VerticesWithEdges < 3K: %ld,"
         "VerticesWithEdges < 2K: %ld, VerticesWithEdges < 1K: %ld, MaxEdgesOfTransit: %d\n", 
         identityKernelTransits, identityKernelSamples, maxEdgesOfIdentityTransits, 
         subWarpLevelTransits, subWarpLevelSamples, maxEdgesOfSubWarpTransits, 
            subWarpTransitsWithEdgesMoreThan384, subWarpTransitsWithEdgesLessThan384, numSubWarps, 
         threadBlockLevelTransits, threadBlockLevelSamples, tbVerticesWithEdgesMoreThan3K, tbVerticesWithEdgesLessThan3K,
         gridLevelTransits, gridLevelSamples, gridVerticesWithEdgesMoreThan3K, gridVerticesWithEdgesLessThan3K, gridVerticesWithEdgesLessThan2K, gridVerticesWithEdgesLessThan1K, maxEdgesOfGridTransits);

  delete hUniqueTransits;
  delete hUniqueTransitsCounts;
  delete hUniqueTransitsNumRuns;
}

template<class SampleType, typename App>
bool doTransitParallelSampling(CSR* csr, NextDoorData<SampleType, App>& nextDoorData, bool enableLoadBalancing)
{
  //Size of each sample output
  size_t maxNeighborsToSample = (App().samplingType() == CollectiveNeighborhood) ? 1 : App().initialSampleSize(csr);
  for (int step = 0; step < App().steps() - 1; step++) {
    if (App().samplingType() == CollectiveNeighborhood) {
      maxNeighborsToSample = max(maxNeighborsToSample, (size_t)App().stepSize(step));
    } else {
      maxNeighborsToSample *= App().stepSize(step);
    }
  }
  std::vector<GPUCSRPartition>& gpuCSRPartitions = nextDoorData.gpuCSRPartitions;

  const size_t numDevices = nextDoorData.devices.size();
  size_t finalSampleSize = getFinalSampleSize<App>();
  for(auto deviceIdx = 0; deviceIdx < nextDoorData.devices.size(); deviceIdx++) {
    auto device = nextDoorData.devices[deviceIdx];
    CHK_CU(hipSetDevice(device));
    const size_t perDeviceNumSamples = PartDivisionSize(nextDoorData.samples.size(), deviceIdx, numDevices);
    const size_t deviceSampleStartPtr = PartStartPointer(nextDoorData.samples.size(), deviceIdx, numDevices);
    if (App().steps() == 1) {
      CHK_CU(hipMemcpy(nextDoorData.dSamplesToTransitMapValues[deviceIdx], 
                        &nextDoorData.initialContents[0] + App().initialSampleSize(csr)*deviceSampleStartPtr, 
                        sizeof(VertexID_t)*App().initialSampleSize(csr)*perDeviceNumSamples, 
                        hipMemcpyHostToDevice));
      CHK_CU(hipMemcpy(nextDoorData.dSamplesToTransitMapKeys[deviceIdx], 
                        &nextDoorData.initialTransitToSampleValues[0] + App().initialSampleSize(csr)*deviceSampleStartPtr, 
                        sizeof(VertexID_t)*App().initialSampleSize(csr)*perDeviceNumSamples, 
                        hipMemcpyHostToDevice));
    } else {
      CHK_CU(hipMemcpy(nextDoorData.dTransitToSampleMapKeys[deviceIdx], 
                        &nextDoorData.initialContents[0] + App().initialSampleSize(csr)*deviceSampleStartPtr, 
                        sizeof(VertexID_t)*App().initialSampleSize(csr)*perDeviceNumSamples, 
                        hipMemcpyHostToDevice));
      CHK_CU(hipMemcpy(nextDoorData.dTransitToSampleMapValues[deviceIdx], 
                        &nextDoorData.initialTransitToSampleValues[0] + App().initialSampleSize(csr)*deviceSampleStartPtr,  
                        sizeof(VertexID_t)*App().initialSampleSize(csr)*perDeviceNumSamples, 
                        hipMemcpyHostToDevice));
    }
  }

  // for (auto v : nextDoorData.initialTransitToSampleValues) {
  //   if (v != 0) {
  //     printf("v %d\n", v);
  //   }
  // }
  std::vector<VertexID_t*> d_temp_storage = std::vector<VertexID_t*>(nextDoorData.devices.size());
  std::vector<size_t> temp_storage_bytes = std::vector<size_t>(nextDoorData.devices.size());;

  std::vector<VertexID_t*> dUniqueTransits = std::vector<VertexID_t*>(nextDoorData.devices.size());
  std::vector<VertexID_t*> dUniqueTransitsCounts = std::vector<VertexID_t*>(nextDoorData.devices.size());
  std::vector<EdgePos_t*> dUniqueTransitsNumRuns = std::vector<EdgePos_t*>(nextDoorData.devices.size());
  std::vector<EdgePos_t*> dTransitPositions = std::vector<EdgePos_t*>(nextDoorData.devices.size());
  std::vector<EdgePos_t*> uniqueTransitNumRuns = std::vector<EdgePos_t*>(nextDoorData.devices.size());
   
  /**Pointers for each kernel type**/
  std::vector<EdgePos_t*> gridKernelTransitsNum = std::vector<EdgePos_t*>(nextDoorData.devices.size());
  std::vector<EdgePos_t*> dGridKernelTransitsNum = std::vector<EdgePos_t*>(nextDoorData.devices.size());
  std::vector<VertexID_t*> dGridKernelTransits = std::vector<VertexID_t*>(nextDoorData.devices.size());
  
  std::vector<EdgePos_t*> threadBlockKernelTransitsNum = std::vector<EdgePos_t*>(nextDoorData.devices.size());
  std::vector<EdgePos_t*> dThreadBlockKernelTransitsNum = std::vector<EdgePos_t*>(nextDoorData.devices.size());
  std::vector<VertexID_t*> dThreadBlockKernelTransits = std::vector<VertexID_t*>(nextDoorData.devices.size());

  std::vector<EdgePos_t*> subWarpKernelTransitsNum = std::vector<EdgePos_t*>(nextDoorData.devices.size());
  std::vector<EdgePos_t*> dSubWarpKernelTransitsNum = std::vector<EdgePos_t*>(nextDoorData.devices.size());
  std::vector<VertexID_t*> dSubWarpKernelTransits = std::vector<VertexID_t*>(nextDoorData.devices.size());

  std::vector<EdgePos_t*> identityKernelTransitsNum = std::vector<EdgePos_t*>(nextDoorData.devices.size());
  std::vector<EdgePos_t*> dIdentityKernelTransitsNum = std::vector<EdgePos_t*>(nextDoorData.devices.size());
  /**********************************/
  
  /****Variables for Collective Transit Sampling***/
  std::vector<EdgePos_t*> hSumNeighborhoodSizes = std::vector<EdgePos_t*>(nextDoorData.devices.size(), nullptr);
  std::vector<EdgePos_t*> dSumNeighborhoodSizes = std::vector<EdgePos_t*>(nextDoorData.devices.size(), nullptr);
  std::vector<EdgePos_t*> dSampleNeighborhoodPos = std::vector<EdgePos_t*>(nextDoorData.devices.size(), nullptr);
  std::vector<EdgePos_t*> dSampleNeighborhoodSizes = std::vector<EdgePos_t*>(nextDoorData.devices.size(), nullptr);
  std::vector<VertexID_t*> dCollectiveNeighborhoodCSRCols = std::vector<VertexID_t*>(nextDoorData.devices.size(), nullptr);
  std::vector<EdgePos_t*> dCollectiveNeighborhoodCSRRows = std::vector<EdgePos_t*>(nextDoorData.devices.size(), nullptr);

  if (App().samplingType() == SamplingType::CollectiveNeighborhood) {
    for(auto idx = 0; idx < nextDoorData.devices.size(); idx++) {
      auto device = nextDoorData.devices[idx];
      CHK_CU(hipSetDevice(device));
      const size_t perDeviceNumSamples = PartDivisionSize(nextDoorData.samples.size(), idx, numDevices);
      CHK_CU(hipHostMalloc(&hSumNeighborhoodSizes[idx], sizeof(EdgePos_t)));
      CHK_CU(hipMalloc(&dSumNeighborhoodSizes[idx], sizeof(EdgePos_t)));
      CHK_CU(hipMalloc(&dSampleNeighborhoodPos[idx], sizeof(EdgePos_t)*perDeviceNumSamples));
      CHK_CU(hipMalloc(&dSampleNeighborhoodSizes[idx], sizeof(EdgePos_t)*perDeviceNumSamples));
      CHK_CU(hipMemset(dSampleNeighborhoodSizes[idx], 0, sizeof(EdgePos_t)*perDeviceNumSamples));
    }
  }

  std::vector<EdgePos_t*> dInvalidVertexStartPosInMap = std::vector<EdgePos_t*>(nextDoorData.devices.size(), nullptr);
  std::vector<EdgePos_t*> invalidVertexStartPosInMap = std::vector<EdgePos_t*>(nextDoorData.devices.size(), nullptr);
  
  /*Single Memory Location on both CPU and GPU for transferring
   *number of transits for all kernels */
  std::vector<EdgePos_t*> dKernelTransitNums = std::vector<EdgePos_t*>(nextDoorData.devices.size(), nullptr);
  std::vector<EdgePos_t*> hKernelTransitNums= std::vector<EdgePos_t*>(nextDoorData.devices.size(), nullptr);
  const int NUM_KERNEL_TYPES = TransitKernelTypes::NumKernelTypes + 1;

  std::vector<int*> dKernelTypeForTransit = std::vector<int*>(nextDoorData.devices.size(), nullptr);;

  for(auto deviceIdx = 0; deviceIdx < nextDoorData.devices.size(); deviceIdx++) {
    auto device = nextDoorData.devices[deviceIdx];
    const size_t perDeviceNumSamples = PartDivisionSize(nextDoorData.samples.size(), deviceIdx, numDevices);
    CHK_CU(hipSetDevice(device));
    CHK_CU(hipHostMalloc(&uniqueTransitNumRuns[deviceIdx], sizeof(EdgePos_t)));
    CHK_CU(hipHostMalloc(&hKernelTransitNums[deviceIdx], NUM_KERNEL_TYPES * sizeof(EdgePos_t)));
    
    gridKernelTransitsNum[deviceIdx] = hKernelTransitNums[deviceIdx];
    threadBlockKernelTransitsNum[deviceIdx] = hKernelTransitNums[deviceIdx] + 1;
    subWarpKernelTransitsNum[deviceIdx] = hKernelTransitNums[deviceIdx] + 2;
    identityKernelTransitsNum[deviceIdx] = hKernelTransitNums[deviceIdx] + 3;
    invalidVertexStartPosInMap[deviceIdx] = hKernelTransitNums[deviceIdx] + 4;
    //threadBlockKernelTransitsNum = hKernelTransitNums[3];
    CHK_CU(hipMalloc(&dKernelTypeForTransit[deviceIdx], sizeof(VertexID_t)*csr->get_n_vertices()));
    CHK_CU(hipMalloc(&dTransitPositions[deviceIdx], 
                      sizeof(VertexID_t)*csr->get_n_vertices()));
    CHK_CU(hipMalloc(&dGridKernelTransits[deviceIdx], 
                      sizeof(VertexID_t)*perDeviceNumSamples*maxNeighborsToSample));
    std::cout << "perDeviceNumSamples " << perDeviceNumSamples << " maxNeighborsToSample " << maxNeighborsToSample << std::endl;
    if (useThreadBlockKernel) {
      CHK_CU(hipMalloc(&dThreadBlockKernelTransits[deviceIdx], 
                      sizeof(VertexID_t)*perDeviceNumSamples*maxNeighborsToSample));
    }

    if (useSubWarpKernel) {
      CHK_CU(hipMalloc(&dSubWarpKernelTransits[deviceIdx],
                      sizeof(VertexID_t)*perDeviceNumSamples*maxNeighborsToSample));
    }

    CHK_CU(hipMalloc(&dKernelTransitNums[deviceIdx], NUM_KERNEL_TYPES * sizeof(EdgePos_t)));
    CHK_CU(hipMemset(dKernelTransitNums[deviceIdx], 0, NUM_KERNEL_TYPES * sizeof(EdgePos_t)));
    dGridKernelTransitsNum[deviceIdx] = dKernelTransitNums[deviceIdx];
    dThreadBlockKernelTransitsNum[deviceIdx] = dKernelTransitNums[deviceIdx] + 1;
    dSubWarpKernelTransitsNum[deviceIdx] = dKernelTransitNums[deviceIdx] + 2;
    dIdentityKernelTransitsNum[deviceIdx] = dKernelTransitNums[deviceIdx] + 3;
    dInvalidVertexStartPosInMap[deviceIdx] = dKernelTransitNums[deviceIdx] + 4;

    //Check if the space runs out.
    //TODO: Use DoubleBuffer version that requires O(P) space.
    hipcub::DeviceRadixSort::SortPairs(d_temp_storage[deviceIdx], temp_storage_bytes[deviceIdx], 
              nextDoorData.dSamplesToTransitMapValues[deviceIdx], nextDoorData.dTransitToSampleMapKeys[deviceIdx], 
              nextDoorData.dSamplesToTransitMapKeys[deviceIdx], nextDoorData.dTransitToSampleMapValues[deviceIdx], 
              perDeviceNumSamples*maxNeighborsToSample);

    CHK_CU(hipMalloc(&d_temp_storage[deviceIdx], temp_storage_bytes[deviceIdx]));
    CHK_CU(hipMemset(nextDoorData.dSampleInsertionPositions[deviceIdx], 0, sizeof(EdgePos_t)*perDeviceNumSamples));

    CHK_CU(hipMalloc(&dUniqueTransits[deviceIdx], (csr->get_n_vertices() + 1)*sizeof(VertexID_t)));
    CHK_CU(hipMalloc(&dUniqueTransitsCounts[deviceIdx], (csr->get_n_vertices() + 1)*sizeof(VertexID_t)));
    CHK_CU(hipMalloc(&dUniqueTransitsNumRuns[deviceIdx], sizeof(size_t)));

    CHK_CU(hipMemset(dUniqueTransitsCounts[deviceIdx], 0, (csr->get_n_vertices() + 1)*sizeof(VertexID_t)));
    CHK_CU(hipMemset(dUniqueTransitsNumRuns[deviceIdx], 0, sizeof(size_t)));
  }

  std::vector<VertexID_t*> hAllSamplesToTransitMapKeys;
  std::vector<VertexID_t*> hAllTransitToSampleMapValues;
  std::vector<size_t> totalTransits = std::vector<size_t>(nextDoorData.devices.size());

  double loadBalancingTime = 0;
  double inversionTime = 0;
  double gridKernelTime = 0;
  double subWarpKernelTime = 0;
  double identityKernelTime = 0;
  double threadBlockKernelTime = 0;
  size_t neighborsToSampleAtStep = (App().samplingType() == CollectiveNeighborhood) ? 1 : App().initialSampleSize(csr);

  double end_to_end_t1 = convertTimeValToDouble(getTimeOfDay ());
  for (int step = 0; step < App().steps(); step++) {
    const size_t numTransits = (App().samplingType() == CollectiveNeighborhood) ? 1 : neighborsToSampleAtStep;
    std::vector<size_t> totalThreads = std::vector<size_t>(nextDoorData.devices.size());
    for(int i = 0; i < nextDoorData.devices.size(); i++) {
      const size_t perDeviceNumSamples = PartDivisionSize(nextDoorData.samples.size(), i, numDevices);
      totalThreads[i] = perDeviceNumSamples*neighborsToSampleAtStep;
    }
    std::cout << "step " << step << std::endl;
    if (App().steps() == 1) {
      //FIXME: Currently a non-sorted Transit to Sample Map is passed to both TP and TP+LB.
      //Here, if there is only one step, a sorted map is passed.
      //Fix this to make sure a sorted map is always passed.
      double inversionT1 = convertTimeValToDouble(getTimeOfDay ());
      for(auto deviceIdx = 0; deviceIdx < nextDoorData.devices.size(); deviceIdx++) {
        auto device = nextDoorData.devices[deviceIdx];
        CHK_CU(hipSetDevice(device));
        //Invert sample->transit map by sorting samples based on the transit vertices
        hipcub::DeviceRadixSort::SortPairs(d_temp_storage[deviceIdx], temp_storage_bytes[deviceIdx], 
                                        nextDoorData.dSamplesToTransitMapValues[deviceIdx], nextDoorData.dTransitToSampleMapKeys[deviceIdx], 
                                        nextDoorData.dSamplesToTransitMapKeys[deviceIdx], nextDoorData.dTransitToSampleMapValues[deviceIdx], 
                                        totalThreads[deviceIdx], 0, nextDoorData.maxBits);
        CHK_CU(hipGetLastError());
      }
      
      CUDA_SYNC_DEVICE_ALL(nextDoorData);
      double inversionT2 = convertTimeValToDouble(getTimeOfDay ());
      //std::cout << "inversionTime at step " << step << " : " << (inversionT2 - inversionT1) << std::endl; 
      inversionTime += (inversionT2 - inversionT1);
    }

    neighborsToSampleAtStep = (App().samplingType() == CollectiveNeighborhood) ? App().stepSize(step) : neighborsToSampleAtStep * App().stepSize(step);    
    for(int i = 0; i < nextDoorData.devices.size(); i++) {
      const size_t perDeviceNumSamples = PartDivisionSize(nextDoorData.samples.size(), i, numDevices);
      totalThreads[i] = perDeviceNumSamples*neighborsToSampleAtStep;
    }

    if ((step == 0 && App().steps() > 1) || !enableLoadBalancing) {
      //When not doing load balancing call baseline transit parallel
      if (App().samplingType() == SamplingType::CollectiveNeighborhood) {
        for(auto deviceIdx = 0; deviceIdx < nextDoorData.devices.size(); deviceIdx++) {
          auto device = nextDoorData.devices[deviceIdx];
          CHK_CU(hipSetDevice(device));

          CHK_CU(hipMemset(nextDoorData.dSampleInsertionPositions[deviceIdx], 0, 
                            sizeof(VertexID_t) * nextDoorData.samples.size()));
          CHK_CU(hipMemset(dSumNeighborhoodSizes[deviceIdx], 0, sizeof(EdgePos_t)));
          //Create collective neighborhood for all transits related to a sample
          collectiveNeighbrsSize<App><<<nextDoorData.samples.size(), N_THREADS>>>(step, gpuCSRPartitions[deviceIdx], 
                                                                              nextDoorData.INVALID_VERTEX,
                                                                              nextDoorData.dInitialSamples[deviceIdx], 
                                                                              nextDoorData.dFinalSamples[deviceIdx], 
                                                                              nextDoorData.samples.size(),
                                                                              dSampleNeighborhoodPos[deviceIdx],
                                                                              dSumNeighborhoodSizes[deviceIdx]);
          CHK_CU(hipGetLastError());
        }

        CUDA_SYNC_DEVICE_ALL(nextDoorData);

        for(auto deviceIdx = 0; deviceIdx < nextDoorData.devices.size(); deviceIdx++) {
          auto device = nextDoorData.devices[deviceIdx];
          CHK_CU(hipSetDevice(device));
          //TODO: Neighborhood is edges of all transit vertices. Hence, neighborhood size is (# of Transit Vertices)/(|G.V|) * |G.E|
          CHK_CU(hipMemcpy(hSumNeighborhoodSizes[deviceIdx], dSumNeighborhoodSizes[deviceIdx], sizeof(EdgePos_t), hipMemcpyDeviceToHost));        
          CHK_CU(hipMalloc(&dCollectiveNeighborhoodCSRCols[deviceIdx], sizeof(VertexID_t)*(*hSumNeighborhoodSizes[deviceIdx])));
          CHK_CU(hipMalloc(&dCollectiveNeighborhoodCSRRows[deviceIdx], sizeof(EdgePos_t)*App().initialSampleSize(csr)*nextDoorData.samples.size()));
        }

        for(auto deviceIdx = 0; deviceIdx < nextDoorData.devices.size(); deviceIdx++) {
          auto device = nextDoorData.devices[deviceIdx];
          CHK_CU(hipSetDevice(device));
          const VertexID_t deviceSampleStartPtr = PartStartPointer(nextDoorData.samples.size(), deviceIdx, numDevices);
          CHK_CU(hipMemset(nextDoorData.dSampleInsertionPositions[deviceIdx], 0, sizeof(VertexID_t) * nextDoorData.samples.size()));
          //Compute collective neighborhood using transit parallel kernel
          for (int threadsExecuted = 0; threadsExecuted < totalThreads[deviceIdx]; threadsExecuted += nextDoorData.maxThreadsPerKernel[deviceIdx]) {
            size_t currExecutionThreads = min((size_t)nextDoorData.maxThreadsPerKernel[deviceIdx], totalThreads[deviceIdx] - threadsExecuted);
            samplingKernel<SampleType, App, TransitParallelMode::CollectiveNeighborhoodComputation, 32><<<thread_block_size(currExecutionThreads, N_THREADS), N_THREADS>>>(step, gpuCSRPartitions[deviceIdx], 
                            threadsExecuted, currExecutionThreads, deviceSampleStartPtr, nextDoorData.INVALID_VERTEX,
                            (const VertexID_t*)nextDoorData.dTransitToSampleMapKeys[deviceIdx], (const VertexID_t*)nextDoorData.dTransitToSampleMapValues[deviceIdx],
                            totalThreads[deviceIdx], nextDoorData.dOutputSamples[deviceIdx], nextDoorData.samples.size(),
                            nextDoorData.dSamplesToTransitMapKeys[deviceIdx], nextDoorData.dSamplesToTransitMapValues[deviceIdx],
                            nextDoorData.dFinalSamples[deviceIdx], finalSampleSize, nextDoorData.dSampleInsertionPositions[deviceIdx],
                            dSampleNeighborhoodSizes[deviceIdx], dSampleNeighborhoodPos[deviceIdx], dCollectiveNeighborhoodCSRRows[deviceIdx], dCollectiveNeighborhoodCSRCols[deviceIdx], 
                            nextDoorData.dCurandStates[deviceIdx]);
            CHK_CU(hipGetLastError());
            // CHK_CU(hipDeviceSynchronize());
          }
        }

        CUDA_SYNC_DEVICE_ALL(nextDoorData);
      } else {
        for(auto deviceIdx = 0; deviceIdx < nextDoorData.devices.size(); deviceIdx++) {
          auto device = nextDoorData.devices[deviceIdx];
          CHK_CU(hipSetDevice(device));
          const VertexID_t deviceSampleStartPtr = PartStartPointer(nextDoorData.samples.size(), deviceIdx, numDevices);
          for (int threadsExecuted = 0; threadsExecuted < totalThreads[deviceIdx]; threadsExecuted += nextDoorData.maxThreadsPerKernel[deviceIdx]) {
            size_t currExecutionThreads = min((size_t)nextDoorData.maxThreadsPerKernel[deviceIdx], totalThreads[deviceIdx] - threadsExecuted);
            samplingKernel<SampleType, App, TransitParallelMode::NextFuncExecution, 0><<<thread_block_size(currExecutionThreads, N_THREADS), N_THREADS>>>(step, gpuCSRPartitions[deviceIdx], 
                            threadsExecuted, currExecutionThreads, deviceSampleStartPtr, nextDoorData.INVALID_VERTEX,
                            (const VertexID_t*)nextDoorData.dTransitToSampleMapKeys[deviceIdx], (const VertexID_t*)nextDoorData.dTransitToSampleMapValues[deviceIdx],
                            totalThreads[deviceIdx], nextDoorData.dOutputSamples[deviceIdx], nextDoorData.samples.size(),
                            nextDoorData.dSamplesToTransitMapKeys[deviceIdx], nextDoorData.dSamplesToTransitMapValues[deviceIdx],
                            nextDoorData.dFinalSamples[deviceIdx], finalSampleSize, nextDoorData.dSampleInsertionPositions[deviceIdx],
                            nullptr,  nullptr,  nullptr,  nullptr, nextDoorData.dCurandStates[deviceIdx]);
            CHK_CU(hipGetLastError());
          }
        }

        CUDA_SYNC_DEVICE_ALL(nextDoorData);
      }
    } else {
      if (App().samplingType() == SamplingType::CollectiveNeighborhood) {
        for(auto deviceIdx = 0; deviceIdx < nextDoorData.devices.size(); deviceIdx++) {
          auto device = nextDoorData.devices[deviceIdx];
          CHK_CU(hipSetDevice(device));
          CHK_CU(hipMemset(nextDoorData.dSampleInsertionPositions[deviceIdx], 0, sizeof(VertexID_t) * nextDoorData.samples.size()));
          CHK_CU(hipMemset(dSumNeighborhoodSizes[deviceIdx], 0, sizeof(EdgePos_t)));
          //Create collective neighborhood for all transits related to a sample
          collectiveNeighbrsSize<App><<<nextDoorData.samples.size(), N_THREADS>>>(step, gpuCSRPartitions[deviceIdx], 
                                                                              nextDoorData.INVALID_VERTEX,
                                                                              nextDoorData.dInitialSamples[deviceIdx], 
                                                                              nextDoorData.dFinalSamples[deviceIdx], 
                                                                              nextDoorData.samples.size(),
                                                                              dSampleNeighborhoodPos[deviceIdx],
                                                                              dSumNeighborhoodSizes[deviceIdx]);
          CHK_CU(hipGetLastError());
          CHK_CU(hipDeviceSynchronize());
          //TODO: Neighborhood is edges of all transit vertices. Hence, neighborhood size is (# of Transit Vertices)/(|G.V|) * |G.E|
          CHK_CU(hipMemcpy(hSumNeighborhoodSizes[deviceIdx], dSumNeighborhoodSizes[deviceIdx], sizeof(EdgePos_t), hipMemcpyDeviceToHost));
          //std::cout <<" hSumNeighborhoodSizes " << *hSumNeighborhoodSizes << std::endl;
          CHK_CU(hipMalloc(&dCollectiveNeighborhoodCSRCols[deviceIdx], sizeof(VertexID_t)*(*hSumNeighborhoodSizes[deviceIdx])));
          CHK_CU(hipMalloc(&dCollectiveNeighborhoodCSRRows[deviceIdx], sizeof(EdgePos_t)*App().initialSampleSize(csr)*nextDoorData.samples.size()));
        }
      } else {
        double loadBalancingT1 = convertTimeValToDouble(getTimeOfDay ());
        
        for(auto deviceIdx = 0; deviceIdx < nextDoorData.devices.size(); deviceIdx++) {
          auto device = nextDoorData.devices[deviceIdx];
          CHK_CU(hipSetDevice(device));
          CHK_CU(hipMemset(dKernelTransitNums[deviceIdx], 0, NUM_KERNEL_TYPES * sizeof(EdgePos_t)));
          CHK_CU(hipMemset(dInvalidVertexStartPosInMap[deviceIdx], 0xFF, sizeof(EdgePos_t)));
          const size_t perDeviceNumSamples = PartDivisionSize(nextDoorData.samples.size(), deviceIdx, numDevices);
          totalTransits[deviceIdx] = perDeviceNumSamples*numTransits;

          //Find the index of first invalid transit vertex. 
          invalidVertexStartPos<<<DIVUP(totalTransits[deviceIdx], 256), 256>>>(step, nextDoorData.dTransitToSampleMapKeys[deviceIdx], 
                                                                               totalTransits[deviceIdx], nextDoorData.INVALID_VERTEX, 
                                                                               dInvalidVertexStartPosInMap[deviceIdx]);
          CHK_CU(hipGetLastError());
        }

        CUDA_SYNC_DEVICE_ALL(nextDoorData);

        for(auto deviceIdx = 0; deviceIdx < nextDoorData.devices.size(); deviceIdx++) {
          auto device = nextDoorData.devices[deviceIdx];
          CHK_CU(hipSetDevice(device));
          CHK_CU(hipMemcpy(invalidVertexStartPosInMap[deviceIdx], dInvalidVertexStartPosInMap[deviceIdx], 
                            1 * sizeof(EdgePos_t), hipMemcpyDeviceToHost));
          //Now the number of threads launched are equal to number of valid transit vertices
          if (*invalidVertexStartPosInMap[deviceIdx] == 0xFFFFFFFF) {
            *invalidVertexStartPosInMap[deviceIdx] = totalTransits[deviceIdx];
          }
          totalThreads[deviceIdx] = *invalidVertexStartPosInMap[deviceIdx];
        }

        for(auto deviceIdx = 0; deviceIdx < nextDoorData.devices.size(); deviceIdx++) {
          auto device = nextDoorData.devices[deviceIdx];
          CHK_CU(hipSetDevice(device));
          void* dRunLengthEncodeTmpStorage = nullptr;
          size_t dRunLengthEncodeTmpStorageSize = 0;
          //Find the number of transit vertices
          hipcub::DeviceRunLengthEncode::Encode(dRunLengthEncodeTmpStorage, dRunLengthEncodeTmpStorageSize, 
                                            nextDoorData.dTransitToSampleMapKeys[deviceIdx],
                                            dUniqueTransits[deviceIdx], dUniqueTransitsCounts[deviceIdx], 
                                            dUniqueTransitsNumRuns[deviceIdx], totalThreads[deviceIdx]);

          assert(dRunLengthEncodeTmpStorageSize < temp_storage_bytes[deviceIdx]);
          dRunLengthEncodeTmpStorage = d_temp_storage[deviceIdx];
          hipcub::DeviceRunLengthEncode::Encode(dRunLengthEncodeTmpStorage, dRunLengthEncodeTmpStorageSize, 
                                            nextDoorData.dTransitToSampleMapKeys[deviceIdx],
                                            dUniqueTransits[deviceIdx], dUniqueTransitsCounts[deviceIdx], 
                                            dUniqueTransitsNumRuns[deviceIdx], totalThreads[deviceIdx]);

          CHK_CU(hipGetLastError());
        }
        
        CUDA_SYNC_DEVICE_ALL(nextDoorData);
        
        for(auto deviceIdx = 0; deviceIdx < nextDoorData.devices.size(); deviceIdx++) {
          auto device = nextDoorData.devices[deviceIdx];
          CHK_CU(hipSetDevice(device));
          CHK_CU(hipMemcpy(uniqueTransitNumRuns[deviceIdx], dUniqueTransitsNumRuns[deviceIdx], 
                            sizeof(*uniqueTransitNumRuns[deviceIdx]), hipMemcpyDeviceToHost));
          void* dExclusiveSumTmpStorage = nullptr;
          size_t dExclusiveSumTmpStorageSize = 0;
          //Exclusive sum to obtain the start position of each transit (and its samples) in the map
          hipcub::DeviceScan::ExclusiveSum(dExclusiveSumTmpStorage, dExclusiveSumTmpStorageSize, dUniqueTransitsCounts[deviceIdx], 
                                        dTransitPositions[deviceIdx], *uniqueTransitNumRuns[deviceIdx]);

          assert(dExclusiveSumTmpStorageSize < temp_storage_bytes[deviceIdx]);
          dExclusiveSumTmpStorage = d_temp_storage[deviceIdx];

          hipcub::DeviceScan::ExclusiveSum(dExclusiveSumTmpStorage, dExclusiveSumTmpStorageSize, dUniqueTransitsCounts[deviceIdx],
                                        dTransitPositions[deviceIdx], *uniqueTransitNumRuns[deviceIdx]);

          CHK_CU(hipGetLastError());
        }

        CUDA_SYNC_DEVICE_ALL(nextDoorData);

        int subWarpSize = subWarpSizeAtStep<App>(step);
        // printKernelTypes<App>(step, csr, dUniqueTransits[0], dUniqueTransitsCounts[0], dUniqueTransitsNumRuns[0]);
        for(auto deviceIdx = 0; deviceIdx < nextDoorData.devices.size(); deviceIdx++) {
          auto device = nextDoorData.devices[deviceIdx];
          CHK_CU(hipSetDevice(device));
          if (*uniqueTransitNumRuns[deviceIdx] == 0) 
            continue;

          partitionTransitsInKernels<App, 1024, TransitKernelTypes::GridKernel, true><<<thread_block_size((*uniqueTransitNumRuns[deviceIdx]), 1024), 1024>>>(step, dUniqueTransits[deviceIdx], dUniqueTransitsCounts[deviceIdx], 
              dTransitPositions[deviceIdx], *uniqueTransitNumRuns[deviceIdx], nextDoorData.INVALID_VERTEX, dGridKernelTransits[deviceIdx], dGridKernelTransitsNum[deviceIdx], 
              dThreadBlockKernelTransits[deviceIdx], dThreadBlockKernelTransitsNum[deviceIdx], dSubWarpKernelTransits[deviceIdx], dSubWarpKernelTransitsNum[deviceIdx], nullptr, 
              dIdentityKernelTransitsNum[deviceIdx], dKernelTypeForTransit[deviceIdx], nextDoorData.dTransitToSampleMapKeys[deviceIdx]);

          CHK_CU(hipGetLastError());
        }

        CUDA_SYNC_DEVICE_ALL(nextDoorData);

        if (useThreadBlockKernel and subWarpSize > 1) {
          for(auto deviceIdx = 0; deviceIdx < nextDoorData.devices.size(); deviceIdx++) {
            auto device = nextDoorData.devices[deviceIdx];
            CHK_CU(hipSetDevice(device));
            if (*uniqueTransitNumRuns[deviceIdx] == 0) 
              continue;

            partitionTransitsInKernels<App, 1024, TransitKernelTypes::ThreadBlockKernel, false><<<thread_block_size((*uniqueTransitNumRuns[deviceIdx]), 1024), 1024>>>(step, dUniqueTransits[deviceIdx], dUniqueTransitsCounts[deviceIdx], 
                dTransitPositions[deviceIdx], *uniqueTransitNumRuns[deviceIdx], nextDoorData.INVALID_VERTEX, dGridKernelTransits[deviceIdx], dGridKernelTransitsNum[deviceIdx], 
                dThreadBlockKernelTransits[deviceIdx], dThreadBlockKernelTransitsNum[deviceIdx], dSubWarpKernelTransits[deviceIdx], dSubWarpKernelTransitsNum[deviceIdx], nullptr, 
                dIdentityKernelTransitsNum[deviceIdx], dKernelTypeForTransit[deviceIdx], nextDoorData.dTransitToSampleMapKeys[deviceIdx]);

            CHK_CU(hipGetLastError());
          }

          CUDA_SYNC_DEVICE_ALL(nextDoorData);
        }

        for(auto deviceIdx = 0; deviceIdx < nextDoorData.devices.size(); deviceIdx++) {
          auto device = nextDoorData.devices[deviceIdx];
          CHK_CU(hipSetDevice(device));
          if (*uniqueTransitNumRuns[deviceIdx] == 0) 
            continue;
          CHK_CU(hipMemcpy(hKernelTransitNums[deviceIdx], dKernelTransitNums[deviceIdx], NUM_KERNEL_TYPES * sizeof(EdgePos_t), hipMemcpyDeviceToHost));
          
          //std::cout << "hInvalidVertexStartPosInMap " << *invalidVertexStartPosInMap << " step " << step << std::endl;
          // GPUUtils::printDeviceArray(dGridKernelTransits, *gridKernelTransitsNum, ',');
          // getchar();
          // std::cout << "SubWarpSize at step " << step << " " << subWarpSize << std::endl;
          //From each Transit we sample stepSize(step) vertices
          totalThreads[deviceIdx] =  totalThreads[deviceIdx] * subWarpSize;
        }
        double loadBalancingT2 = convertTimeValToDouble(getTimeOfDay ());
        loadBalancingTime += (loadBalancingT2 - loadBalancingT1);

        bool noTransitsForAllDevices = true;
        for(auto deviceIdx = 0; deviceIdx < nextDoorData.devices.size(); deviceIdx++) {
          auto device = nextDoorData.devices[deviceIdx];
          if (*uniqueTransitNumRuns[deviceIdx] > 0) {
            noTransitsForAllDevices = false;
          }
        }
        if (noTransitsForAllDevices)
          //End Sampling because no more transits exists  
          break;

        double identityKernelTimeT1 = convertTimeValToDouble(getTimeOfDay ());
        for(auto deviceIdx = 0; deviceIdx < nextDoorData.devices.size(); deviceIdx++) {
          auto device = nextDoorData.devices[deviceIdx];
          CHK_CU(hipSetDevice(device));
          if (*uniqueTransitNumRuns[deviceIdx] == 0) 
            continue;
          const size_t maxThreadBlocksPerKernel = min(8192L, nextDoorData.maxThreadsPerKernel[deviceIdx]/256L);
          const VertexID_t deviceSampleStartPtr = PartStartPointer(nextDoorData.samples.size(), deviceIdx, numDevices);
          if (*identityKernelTransitsNum[deviceIdx] > 0) {
            if (App().hasExplicitTransits()) {
              identityKernel<SampleType, App, 256, true, true><<<maxThreadBlocksPerKernel, 256>>>(step, 
                gpuCSRPartitions[deviceIdx], deviceSampleStartPtr, nextDoorData.INVALID_VERTEX,
                (const VertexID_t*)nextDoorData.dTransitToSampleMapKeys[deviceIdx], (const VertexID_t*)nextDoorData.dTransitToSampleMapValues[deviceIdx],
                totalThreads[deviceIdx], nextDoorData.dOutputSamples[deviceIdx], nextDoorData.samples.size(),
                nextDoorData.dSamplesToTransitMapKeys[deviceIdx], nextDoorData.dSamplesToTransitMapValues[deviceIdx],
                nextDoorData.dFinalSamples[deviceIdx], finalSampleSize, nextDoorData.dSampleInsertionPositions[deviceIdx],
                nextDoorData.dCurandStates[deviceIdx], dKernelTypeForTransit[deviceIdx], numberOfTransits<App>(step));
            } else {
              identityKernel<SampleType, App, 256, true, false><<<maxThreadBlocksPerKernel, 256>>>(step, 
                gpuCSRPartitions[deviceIdx], deviceSampleStartPtr, nextDoorData.INVALID_VERTEX,
                (const VertexID_t*)nextDoorData.dTransitToSampleMapKeys[deviceIdx], (const VertexID_t*)nextDoorData.dTransitToSampleMapValues[deviceIdx],
                totalThreads[deviceIdx], nextDoorData.dOutputSamples[deviceIdx], nextDoorData.samples.size(),
                nextDoorData.dSamplesToTransitMapKeys[deviceIdx], nextDoorData.dSamplesToTransitMapValues[deviceIdx],
                nextDoorData.dFinalSamples[deviceIdx], finalSampleSize, nextDoorData.dSampleInsertionPositions[deviceIdx],
                nextDoorData.dCurandStates[deviceIdx], dKernelTypeForTransit[deviceIdx], numberOfTransits<App>(step));
            }
            CHK_CU(hipGetLastError());
          }
        }

        CUDA_SYNC_DEVICE_ALL(nextDoorData);

        double identityKernelTimeT2 = convertTimeValToDouble(getTimeOfDay ());
        identityKernelTime += (identityKernelTimeT2 - identityKernelTimeT1);

        // for(auto deviceIdx = 0; deviceIdx < nextDoorData.devices.size(); deviceIdx++) {
        //   auto device = nextDoorData.devices[deviceIdx];
        //   CHK_CU(hipSetDevice(device));
        //   const int perThreadSamplesForSubWarpKernel = 1;
        //   int threadBlocks = DIVUP(DIVUP(*subWarpKernelTransitsNum[deviceIdx]*LoadBalancing::LoadBalancingThreshold::SubWarpLevel, perThreadSamplesForSubWarpKernel), 256);
        //   //std::cout << "subWarpKernelTransitsNum " << *subWarpKernelTransitsNum << " threadBlocks " << threadBlocks << std::endl;
        //   double subWarpKernelTimeT1 = convertTimeValToDouble(getTimeOfDay ());
        //   if (useSubWarpKernel && *subWarpKernelTransitsNum[deviceIdx] > 0) {
        //     subWarpKernel<SampleType, App, 256,3*1024,false,false,false,perThreadSamplesForSubWarpKernel,true><<<threadBlocks, 256>>>(step, gpuCSRPartition, nextDoorData.INVALID_VERTEX,
        //       (const VertexID_t*)nextDoorData.dTransitToSampleMapKeys[deviceIdx], (const VertexID_t*)nextDoorData.dTransitToSampleMapValues[deviceIdx],
        //       totalThreads[deviceIdx], nextDoorData.dOutputSamples[deviceIdx], nextDoorData.samples.size(),
        //       nextDoorData.dSamplesToTransitMapKeys[deviceIdx], nextDoorData.dSamplesToTransitMapValues[deviceIdx],
        //       nextDoorData.dFinalSamples[deviceIdx], finalSampleSize, nextDoorData.dSampleInsertionPositions[deviceIdx],
        //       nextDoorData.dCurandStates[deviceIdx], dKernelTypeForTransit[deviceIdx], dSubWarpKernelTransits[deviceIdx], *subWarpKernelTransitsNum[deviceIdx]);
        //     CHK_CU(hipGetLastError());
        //     CHK_CU(hipDeviceSynchronize());
        //   }
        //   double subWarpKernelTimeT2 = convertTimeValToDouble(getTimeOfDay ());
        //   subWarpKernelTime += (subWarpKernelTimeT2 - subWarpKernelTimeT1);

        //   double threadBlockKernelTimeT1 = convertTimeValToDouble(getTimeOfDay ());
        //   const int perThreadSamplesForThreadBlockKernel = 1;
        //   threadBlocks = DIVUP(*threadBlockKernelTransitsNum[deviceIdx], perThreadSamplesForThreadBlockKernel);
        //   if (useThreadBlockKernel && *threadBlockKernelTransitsNum[deviceIdx] > 0) {
        //     // threadBlockKernel<SampleType, App, 256,3*1024-3,true,false,false,perThreadSamplesForThreadBlockKernel,true><<<threadBlocks, 32>>>(step, gpuCSRPartition, nextDoorData.INVALID_VERTEX,
        //     //   (const VertexID_t*)nextDoorData.dTransitToSampleMapKeys[deviceIdx], (const VertexID_t*)nextDoorData.dTransitToSampleMapValues[deviceIdx],
        //     //   totalThreads[deviceIdx], nextDoorData.dOutputSamples[deviceIdx], nextDoorData.samples.size(),
        //     //   nextDoorData.dSamplesToTransitMapKeys[deviceIdx], nextDoorData.dSamplesToTransitMapValues[deviceIdx],
        //     //   nextDoorData.dFinalSamples[deviceIdx], finalSampleSize, nextDoorData.dSampleInsertionPositions[deviceIdx],
        //     //   nextDoorData.dCurandStates[deviceIdx], dKernelTypeForTransit[deviceIdx], dThreadBlockKernelTransits[deviceIdx], *threadBlockKernelTransitsNum[deviceIdx]);
        //     // CHK_CU(hipGetLastError());
        //     CHK_CU(hipDeviceSynchronize());
        //   }
        //   
        // }

        if (subWarpSize > 1) {
          EdgePos_t finalSampleSizeTillPreviousStep = 0;
          EdgePos_t neighborsToSampleAtStep = 1;
          for (int _s = 0; _s < step; _s++) {
            neighborsToSampleAtStep *= App().stepSize(_s);
            finalSampleSizeTillPreviousStep += neighborsToSampleAtStep;
          }

          double threadBlockKernelTimeT1 = convertTimeValToDouble(getTimeOfDay ());

          for(auto deviceIdx = 0; deviceIdx < nextDoorData.devices.size(); deviceIdx++) {
            auto device = nextDoorData.devices[deviceIdx];
            CHK_CU(hipSetDevice(device));
            //Process more than one thread blocks positions written in dGridKernelTransits per thread block.
            //Processing more can improve the locality if thread blocks have common transits.
            const int perThreadSamplesForThreadBlockKernel = 8; // Works best for KHop
            const int tbSize = 256L;
            const size_t maxThreadBlocksPerKernel = min(4096L, nextDoorData.maxThreadsPerKernel[deviceIdx]/tbSize);
            const VertexID_t deviceSampleStartPtr = PartStartPointer(nextDoorData.samples.size(), deviceIdx, numDevices);
            const size_t threadBlocks = DIVUP(((*threadBlockKernelTransitsNum[deviceIdx] * LoadBalancing::LoadBalancingThreshold::BlockLevel)/tbSize), perThreadSamplesForThreadBlockKernel);
            if (useThreadBlockKernel && *threadBlockKernelTransitsNum[deviceIdx] > 0){// && numberOfTransits<App>(step) > 1) {
              //FIXME: A Bug in Grid Kernel prevents it from being used when numberOfTransits for a sample at step are 1.
              // for (int threadBlocksExecuted = 0; threadBlocksExecuted < threadBlocks; threadBlocksExecuted += nextDoorData.maxThreadsPerKernel/256) {
                const bool CACHE_EDGES = true;
                const bool CACHE_WEIGHTS = false;
                const int CACHE_SIZE = (CACHE_EDGES || CACHE_WEIGHTS) ? 384 : 0;
                // printf("device %d threadBlockKernelTransitsNum %d threadBlocks %d\n", device, *threadBlockKernelTransitsNum[deviceIdx], threadBlocks);
                switch (subWarpSizeAtStep<App>(step)) {
                  case 32:
                    threadBlockKernel<SampleType,App,tbSize,CACHE_SIZE,CACHE_EDGES,CACHE_WEIGHTS,perThreadSamplesForThreadBlockKernel,false,0,32><<<maxThreadBlocksPerKernel, tbSize>>>(step,
                      gpuCSRPartitions[deviceIdx], deviceSampleStartPtr, nextDoorData.INVALID_VERTEX,
                      (const VertexID_t*)nextDoorData.dTransitToSampleMapKeys[deviceIdx], (const VertexID_t*)nextDoorData.dTransitToSampleMapValues[deviceIdx],
                      totalThreads[deviceIdx],  nextDoorData.dOutputSamples[deviceIdx], nextDoorData.samples.size(),
                      nextDoorData.dSamplesToTransitMapKeys[deviceIdx], nextDoorData.dSamplesToTransitMapValues[deviceIdx],
                      nextDoorData.dFinalSamples[deviceIdx], finalSampleSize, nextDoorData.dSampleInsertionPositions[deviceIdx],
                      nextDoorData.dCurandStates[deviceIdx], dKernelTypeForTransit[deviceIdx], dThreadBlockKernelTransits[deviceIdx], *threadBlockKernelTransitsNum[deviceIdx], threadBlocks,  numberOfTransits<App>(step), finalSampleSizeTillPreviousStep);
                      break;
                  // case 16:
                  //   gridKernel<SampleType,App,256,CACHE_SIZE,CACHE_EDGES,CACHE_WEIGHTS,false,perThreadSamplesForGridKernel,true,true,256,16><<<maxThreadBlocksPerKernel, 256>>>(step,
                  //     gpuCSRPartition, deviceSampleStartPtr, nextDoorData.INVALID_VERTEX,
                  //     (const VertexID_t*)nextDoorData.dTransitToSampleMapKeys[deviceIdx], (const VertexID_t*)nextDoorData.dTransitToSampleMapValues[deviceIdx],
                  //     totalThreads[deviceIdx],  nextDoorData.dOutputSamples[deviceIdx], nextDoorData.samples.size(),
                  //     nextDoorData.dSamplesToTransitMapKeys[deviceIdx], nextDoorData.dSamplesToTransitMapValues[deviceIdx],
                  //     nextDoorData.dFinalSamples[deviceIdx], finalSampleSize, nextDoorData.dSampleInsertionPositions[deviceIdx],
                  //     nextDoorData.dCurandStates[deviceIdx], dKernelTypeForTransit[deviceIdx], dGridKernelTransits[deviceIdx], *gridKernelTransitsNum[deviceIdx], threadBlocks);
                  //     break;
                  // case 8:
                  // gridKernel<SampleType,App,256,CACHE_SIZE,CACHE_EDGES,CACHE_WEIGHTS,false,perThreadSamplesForGridKernel,true,true,256,8><<<maxThreadBlocksPerKernel, 256>>>(step,
                  //   gpuCSRPartition, deviceSampleStartPtr, nextDoorData.INVALID_VERTEX,
                  //     (const VertexID_t*)nextDoorData.dTransitToSampleMapKeys[deviceIdx], (const VertexID_t*)nextDoorData.dTransitToSampleMapValues[deviceIdx],
                  //     totalThreads[deviceIdx],  nextDoorData.dOutputSamples[deviceIdx], nextDoorData.samples.size(),
                  //     nextDoorData.dSamplesToTransitMapKeys[deviceIdx], nextDoorData.dSamplesToTransitMapValues[deviceIdx],
                  //     nextDoorData.dFinalSamples[deviceIdx], finalSampleSize, nextDoorData.dSampleInsertionPositions[deviceIdx],
                  //     nextDoorData.dCurandStates[deviceIdx], dKernelTypeForTransit[deviceIdx], dGridKernelTransits[deviceIdx], *gridKernelTransitsNum[deviceIdx], threadBlocks);
                  //   break;
                  // case 4:
                  // gridKernel<SampleType,App,256,CACHE_SIZE,CACHE_EDGES,CACHE_WEIGHTS,false,perThreadSamplesForGridKernel,true,true,256,4><<<maxThreadBlocksPerKernel, 256>>>(step,
                  //   gpuCSRPartition, deviceSampleStartPtr, nextDoorData.INVALID_VERTEX,
                  //     (const VertexID_t*)nextDoorData.dTransitToSampleMapKeys[deviceIdx], (const VertexID_t*)nextDoorData.dTransitToSampleMapValues[deviceIdx],
                  //     totalThreads[deviceIdx],  nextDoorData.dOutputSamples[deviceIdx], nextDoorData.samples.size(),
                  //     nextDoorData.dSamplesToTransitMapKeys[deviceIdx], nextDoorData.dSamplesToTransitMapValues[deviceIdx],
                  //     nextDoorData.dFinalSamples[deviceIdx], finalSampleSize, nextDoorData.dSampleInsertionPositions[deviceIdx],
                  //     nextDoorData.dCurandStates[deviceIdx], dKernelTypeForTransit[deviceIdx], dGridKernelTransits[deviceIdx], *gridKernelTransitsNum[deviceIdx], threadBlocks);
                  //   break;
                  // case 2:
                  // gridKernel<SampleType,App,256,CACHE_SIZE,CACHE_EDGES,CACHE_WEIGHTS,false,perThreadSamplesForGridKernel,true,true,256,2><<<maxThreadBlocksPerKernel, 256>>>(step,
                  //   gpuCSRPartition, deviceSampleStartPtr, nextDoorData.INVALID_VERTEX,
                  //     (const VertexID_t*)nextDoorData.dTransitToSampleMapKeys[deviceIdx], (const VertexID_t*)nextDoorData.dTransitToSampleMapValues[deviceIdx],
                  //     totalThreads[deviceIdx],  nextDoorData.dOutputSamples[deviceIdx], nextDoorData.samples.size(),
                  //     nextDoorData.dSamplesToTransitMapKeys[deviceIdx], nextDoorData.dSamplesToTransitMapValues[deviceIdx],
                  //     nextDoorData.dFinalSamples[deviceIdx], finalSampleSize, nextDoorData.dSampleInsertionPositions[deviceIdx],
                  //     nextDoorData.dCurandStates[deviceIdx], dKernelTypeForTransit[deviceIdx], dGridKernelTransits[deviceIdx], *gridKernelTransitsNum[deviceIdx], threadBlocks);
                  //   break;
                  // case 1:
                  // gridKernel<SampleType,App,256,CACHE_SIZE,CACHE_EDGES,CACHE_WEIGHTS,false,perThreadSamplesForGridKernel,true,true,256,1><<<maxThreadBlocksPerKernel, 256>>>(step,
                  //   gpuCSRPartition, deviceSampleStartPtr, nextDoorData.INVALID_VERTEX,
                  //     (const VertexID_t*)nextDoorData.dTransitToSampleMapKeys[deviceIdx], (const VertexID_t*)nextDoorData.dTransitToSampleMapValues[deviceIdx],
                  //     totalThreads[deviceIdx],  nextDoorData.dOutputSamples[deviceIdx], nextDoorData.samples.size(),
                  //     nextDoorData.dSamplesToTransitMapKeys[deviceIdx], nextDoorData.dSamplesToTransitMapValues[deviceIdx],
                  //     nextDoorData.dFinalSamples[deviceIdx], finalSampleSize, nextDoorData.dSampleInsertionPositions[deviceIdx],
                  //     nextDoorData.dCurandStates[deviceIdx], dKernelTypeForTransit[deviceIdx], dGridKernelTransits[deviceIdx], *gridKernelTransitsNum[deviceIdx], threadBlocks);
                  //   break;
                  // default:
                  //   //TODO: Add others
                  //     break;
                }
                CHK_CU(hipGetLastError());
                // CHK_CU(hipDeviceSynchronize());
              // }
            }
          }

          CUDA_SYNC_DEVICE_ALL(nextDoorData);

          double threadBlockKernelTimeT2 = convertTimeValToDouble(getTimeOfDay ());
          threadBlockKernelTime += (threadBlockKernelTimeT2 - threadBlockKernelTimeT1);
          double gridKernelTimeT1 = convertTimeValToDouble(getTimeOfDay ());

          for(auto deviceIdx = 0; deviceIdx < nextDoorData.devices.size(); deviceIdx++) {
            auto device = nextDoorData.devices[deviceIdx];
            CHK_CU(hipSetDevice(device));
            //Process more than one thread blocks positions written in dGridKernelTransits per thread block.
            //Processing more can improve the locality if thread blocks have common transits.
            const int perThreadSamplesForGridKernel = 16; // Works best for KHop
            //const int perThreadSamplesForGridKernel = 8;
            
            const size_t maxThreadBlocksPerKernel = min(4096L, nextDoorData.maxThreadsPerKernel[deviceIdx]/256L);
            const VertexID_t deviceSampleStartPtr = PartStartPointer(nextDoorData.samples.size(), deviceIdx, numDevices);
            const size_t threadBlocks = DIVUP(*gridKernelTransitsNum[deviceIdx], perThreadSamplesForGridKernel);
            // printf("device %d gridTransitsNum %d threadBlocks %d\n", device, *gridKernelTransitsNum[deviceIdx], threadBlocks);

            if (useGridKernel && *gridKernelTransitsNum[deviceIdx] > 0){// && numberOfTransits<App>(step) > 1) {
              //FIXME: A Bug in Grid Kernel prevents it from being used when numberOfTransits for a sample at step are 1.
              // for (int threadBlocksExecuted = 0; threadBlocksExecuted < threadBlocks; threadBlocksExecuted += nextDoorData.maxThreadsPerKernel/256) {
                const bool CACHE_EDGES = true;
                const bool CACHE_WEIGHTS = false;
                const int CACHE_SIZE = (CACHE_EDGES || CACHE_WEIGHTS) ? 3*1024-10 : 0;
              
                switch (subWarpSizeAtStep<App>(step)) {
                  case 32:
                    gridKernel<SampleType,App,256,CACHE_SIZE,CACHE_EDGES,CACHE_WEIGHTS,false,perThreadSamplesForGridKernel,true,false,256,32><<<maxThreadBlocksPerKernel, 256>>>(step,
                      gpuCSRPartitions[deviceIdx], deviceSampleStartPtr, nextDoorData.INVALID_VERTEX,
                      (const VertexID_t*)nextDoorData.dTransitToSampleMapKeys[deviceIdx], (const VertexID_t*)nextDoorData.dTransitToSampleMapValues[deviceIdx],
                      totalThreads[deviceIdx],  nextDoorData.dOutputSamples[deviceIdx], nextDoorData.samples.size(),
                      nextDoorData.dSamplesToTransitMapKeys[deviceIdx], nextDoorData.dSamplesToTransitMapValues[deviceIdx],
                      nextDoorData.dFinalSamples[deviceIdx], finalSampleSize, nextDoorData.dSampleInsertionPositions[deviceIdx],
                      nextDoorData.dCurandStates[deviceIdx], dKernelTypeForTransit[deviceIdx], dGridKernelTransits[deviceIdx], 
                      *gridKernelTransitsNum[deviceIdx], threadBlocks,numberOfTransits<App>(step), finalSampleSizeTillPreviousStep);
                      break;
                  case 16:
                    gridKernel<SampleType,App,256,CACHE_SIZE,CACHE_EDGES,CACHE_WEIGHTS,false,perThreadSamplesForGridKernel,true,true,256,16><<<maxThreadBlocksPerKernel, 256>>>(step,
                      gpuCSRPartitions[deviceIdx], deviceSampleStartPtr, nextDoorData.INVALID_VERTEX,
                      (const VertexID_t*)nextDoorData.dTransitToSampleMapKeys[deviceIdx], (const VertexID_t*)nextDoorData.dTransitToSampleMapValues[deviceIdx],
                      totalThreads[deviceIdx],  nextDoorData.dOutputSamples[deviceIdx], nextDoorData.samples.size(),
                      nextDoorData.dSamplesToTransitMapKeys[deviceIdx], nextDoorData.dSamplesToTransitMapValues[deviceIdx],
                      nextDoorData.dFinalSamples[deviceIdx], finalSampleSize, nextDoorData.dSampleInsertionPositions[deviceIdx],
                      nextDoorData.dCurandStates[deviceIdx], dKernelTypeForTransit[deviceIdx], dGridKernelTransits[deviceIdx], *gridKernelTransitsNum[deviceIdx], threadBlocks,numberOfTransits<App>(step), finalSampleSizeTillPreviousStep);
                      break;
                  case 8:
                  gridKernel<SampleType,App,256,CACHE_SIZE,CACHE_EDGES,CACHE_WEIGHTS,false,perThreadSamplesForGridKernel,true,true,256,8><<<maxThreadBlocksPerKernel, 256>>>(step,
                    gpuCSRPartitions[deviceIdx], deviceSampleStartPtr, nextDoorData.INVALID_VERTEX,
                      (const VertexID_t*)nextDoorData.dTransitToSampleMapKeys[deviceIdx], (const VertexID_t*)nextDoorData.dTransitToSampleMapValues[deviceIdx],
                      totalThreads[deviceIdx],  nextDoorData.dOutputSamples[deviceIdx], nextDoorData.samples.size(),
                      nextDoorData.dSamplesToTransitMapKeys[deviceIdx], nextDoorData.dSamplesToTransitMapValues[deviceIdx],
                      nextDoorData.dFinalSamples[deviceIdx], finalSampleSize, nextDoorData.dSampleInsertionPositions[deviceIdx],
                      nextDoorData.dCurandStates[deviceIdx], dKernelTypeForTransit[deviceIdx], dGridKernelTransits[deviceIdx], *gridKernelTransitsNum[deviceIdx], threadBlocks,numberOfTransits<App>(step), finalSampleSizeTillPreviousStep);
                    break;
                  case 4:
                  gridKernel<SampleType,App,256,CACHE_SIZE,CACHE_EDGES,CACHE_WEIGHTS,false,perThreadSamplesForGridKernel,true,true,256,4><<<maxThreadBlocksPerKernel, 256>>>(step,
                    gpuCSRPartitions[deviceIdx], deviceSampleStartPtr, nextDoorData.INVALID_VERTEX,
                      (const VertexID_t*)nextDoorData.dTransitToSampleMapKeys[deviceIdx], (const VertexID_t*)nextDoorData.dTransitToSampleMapValues[deviceIdx],
                      totalThreads[deviceIdx],  nextDoorData.dOutputSamples[deviceIdx], nextDoorData.samples.size(),
                      nextDoorData.dSamplesToTransitMapKeys[deviceIdx], nextDoorData.dSamplesToTransitMapValues[deviceIdx],
                      nextDoorData.dFinalSamples[deviceIdx], finalSampleSize, nextDoorData.dSampleInsertionPositions[deviceIdx],
                      nextDoorData.dCurandStates[deviceIdx], dKernelTypeForTransit[deviceIdx], dGridKernelTransits[deviceIdx], *gridKernelTransitsNum[deviceIdx], threadBlocks,numberOfTransits<App>(step), finalSampleSizeTillPreviousStep);
                    break;
                  case 2:
                  gridKernel<SampleType,App,256,CACHE_SIZE,CACHE_EDGES,CACHE_WEIGHTS,false,perThreadSamplesForGridKernel,true,true,256,2><<<maxThreadBlocksPerKernel, 256>>>(step,
                    gpuCSRPartitions[deviceIdx], deviceSampleStartPtr, nextDoorData.INVALID_VERTEX,
                      (const VertexID_t*)nextDoorData.dTransitToSampleMapKeys[deviceIdx], (const VertexID_t*)nextDoorData.dTransitToSampleMapValues[deviceIdx],
                      totalThreads[deviceIdx],  nextDoorData.dOutputSamples[deviceIdx], nextDoorData.samples.size(),
                      nextDoorData.dSamplesToTransitMapKeys[deviceIdx], nextDoorData.dSamplesToTransitMapValues[deviceIdx],
                      nextDoorData.dFinalSamples[deviceIdx], finalSampleSize, nextDoorData.dSampleInsertionPositions[deviceIdx],
                      nextDoorData.dCurandStates[deviceIdx], dKernelTypeForTransit[deviceIdx], dGridKernelTransits[deviceIdx], *gridKernelTransitsNum[deviceIdx], threadBlocks,numberOfTransits<App>(step), finalSampleSizeTillPreviousStep);
                    break;
                  case 1:
                  gridKernel<SampleType,App,256,CACHE_SIZE,CACHE_EDGES,CACHE_WEIGHTS,false,perThreadSamplesForGridKernel,true,true,256,1><<<maxThreadBlocksPerKernel, 256>>>(step,
                    gpuCSRPartitions[deviceIdx], deviceSampleStartPtr, nextDoorData.INVALID_VERTEX,
                      (const VertexID_t*)nextDoorData.dTransitToSampleMapKeys[deviceIdx], (const VertexID_t*)nextDoorData.dTransitToSampleMapValues[deviceIdx],
                      totalThreads[deviceIdx],  nextDoorData.dOutputSamples[deviceIdx], nextDoorData.samples.size(),
                      nextDoorData.dSamplesToTransitMapKeys[deviceIdx], nextDoorData.dSamplesToTransitMapValues[deviceIdx],
                      nextDoorData.dFinalSamples[deviceIdx], finalSampleSize, nextDoorData.dSampleInsertionPositions[deviceIdx],
                      nextDoorData.dCurandStates[deviceIdx], dKernelTypeForTransit[deviceIdx], dGridKernelTransits[deviceIdx], *gridKernelTransitsNum[deviceIdx], threadBlocks,numberOfTransits<App>(step), finalSampleSizeTillPreviousStep);
                    break;
                  default:
                    //TODO: Add others
                      break;
                }
                CHK_CU(hipGetLastError());
              // }
            }
          }

          CUDA_SYNC_DEVICE_ALL(nextDoorData);

          double gridKernelTimeT2 = convertTimeValToDouble(getTimeOfDay ());
          gridKernelTime += (gridKernelTimeT2 - gridKernelTimeT1);
        }
      }
    }

    if (App().samplingType() == SamplingType::CollectiveNeighborhood) {
      //Call SampleParallel Kernel to do sampling from collective neighborhood
      for(auto deviceIdx = 0; deviceIdx < nextDoorData.devices.size(); deviceIdx++) {
        auto device = nextDoorData.devices[deviceIdx];
        CHK_CU(hipSetDevice(device));

        sampleParallelKernel<SampleType, App, 256, false><<<min(1024L, nextDoorData.maxThreadsPerKernel[deviceIdx]/256L), 256>>>(step, gpuCSRPartitions[deviceIdx], 0,
                    nextDoorData.INVALID_VERTEX, totalThreads[deviceIdx], 
                    nextDoorData.dInitialSamples[deviceIdx], nextDoorData.dOutputSamples[deviceIdx], nextDoorData.samples.size(),
                    nextDoorData.dFinalSamples[deviceIdx], finalSampleSize, 
                    nextDoorData.dSamplesToTransitMapKeys[deviceIdx],
                    nextDoorData.dSamplesToTransitMapValues[deviceIdx],
                    nextDoorData.dSampleInsertionPositions[deviceIdx], nextDoorData.dCurandStates[deviceIdx]);
        CHK_CU(hipGetLastError());
      }

      CUDA_SYNC_DEVICE_ALL(nextDoorData);
      for(auto deviceIdx = 0; deviceIdx < nextDoorData.devices.size(); deviceIdx++) {
        auto device = nextDoorData.devices[deviceIdx];
        CHK_CU(hipSetDevice(device));
        CHK_CU(hipFree(dCollectiveNeighborhoodCSRCols[deviceIdx]));
        CHK_CU(hipFree(dCollectiveNeighborhoodCSRRows[deviceIdx]));
      }
    }
    if (step != App().steps() - 1) {
      double inversionT1 = convertTimeValToDouble(getTimeOfDay ());
      for(auto deviceIdx = 0; deviceIdx < nextDoorData.devices.size(); deviceIdx++) {
        auto device = nextDoorData.devices[deviceIdx];
        CHK_CU(hipSetDevice(device));
        //Invert sample->transit map by sorting samples based on the transit vertices
        hipcub::DeviceRadixSort::SortPairs(d_temp_storage[deviceIdx], temp_storage_bytes[deviceIdx], 
                                        nextDoorData.dSamplesToTransitMapValues[deviceIdx], nextDoorData.dTransitToSampleMapKeys[deviceIdx], 
                                        nextDoorData.dSamplesToTransitMapKeys[deviceIdx], nextDoorData.dTransitToSampleMapValues[deviceIdx], 
                                        totalThreads[deviceIdx], 0, nextDoorData.maxBits);
        CHK_CU(hipGetLastError());
      }
      CUDA_SYNC_DEVICE_ALL(nextDoorData);
      double inversionT2 = convertTimeValToDouble(getTimeOfDay ());
      //std::cout << "inversionTime at step " << step << " : " << (inversionT2 - inversionT1) << std::endl; 
      inversionTime += (inversionT2 - inversionT1);
      #if 0
      VertexID_t* hTransitToSampleMapKeys = new VertexID_t[totalThreads[0]];
      VertexID_t* hTransitToSampleMapValues = new VertexID_t[totalThreads[0]];
      VertexID_t* hSampleToTransitMapKeys = new VertexID_t[totalThreads[0]];
      VertexID_t* hSampleToTransitMapValues = new VertexID_t[totalThreads[0]];

      
      CHK_CU(hipMemcpy(hSampleToTransitMapKeys, nextDoorData.dSamplesToTransitMapKeys[0], 
        totalThreads[0]*sizeof(VertexID_t), hipMemcpyDeviceToHost));
      CHK_CU(hipMemcpy(hSampleToTransitMapValues, nextDoorData.dSamplesToTransitMapValues[0],
        totalThreads[0]*sizeof(VertexID_t), hipMemcpyDeviceToHost));
      CHK_CU(hipMemcpy(hTransitToSampleMapKeys, nextDoorData.dTransitToSampleMapKeys[0], 
                        totalThreads[0]*sizeof(VertexID_t), hipMemcpyDeviceToHost));
      CHK_CU(hipMemcpy(hTransitToSampleMapValues, nextDoorData.dTransitToSampleMapValues[0],
                        totalThreads[0]*sizeof(VertexID_t), hipMemcpyDeviceToHost));
      hAllTransitToSampleMapValues.push_back(hTransitToSampleMapValues);
      hAllSamplesToTransitMapKeys.push_back(hSampleToTransitMapKeys);

      printKeyValuePairs(hTransitToSampleMapKeys, hTransitToSampleMapValues, totalThreads[0], ',');
      #endif
    }
  }

  double end_to_end_t2 = convertTimeValToDouble(getTimeOfDay ());

  std::cout << "Transit Parallel: End to end time " << (end_to_end_t2 - end_to_end_t1) << " secs" << std::endl;
  std::cout << "InversionTime: " << inversionTime <<", " << "LoadBalancingTime: " << loadBalancingTime << ", " << "GridKernelTime: " << gridKernelTime << ", ThreadBlockKernelTime: " << threadBlockKernelTime << ", SubWarpKernelTime: " << subWarpKernelTime << ", IdentityKernelTime: "<< identityKernelTime << std::endl;
  for(auto deviceIdx = 0; deviceIdx < nextDoorData.devices.size(); deviceIdx++) {
    auto device = nextDoorData.devices[deviceIdx];
    CHK_CU(hipFree(d_temp_storage[deviceIdx]));
    if (App().samplingType() == SamplingType::CollectiveNeighborhood) {
      CHK_CU(hipHostFree(hSumNeighborhoodSizes[deviceIdx]));
      CHK_CU(hipFree(dSumNeighborhoodSizes[deviceIdx]));
      CHK_CU(hipFree(dSampleNeighborhoodPos[deviceIdx]));
    }


    CHK_CU(hipFree(dUniqueTransits[deviceIdx]));
    CHK_CU(hipFree(dUniqueTransitsCounts[deviceIdx]));
    CHK_CU(hipFree(dUniqueTransitsNumRuns[deviceIdx]));
    CHK_CU(hipFree(dKernelTypeForTransit[deviceIdx]));
    CHK_CU(hipFree(dTransitPositions[deviceIdx]));
    CHK_CU(hipFree(dGridKernelTransits[deviceIdx]));
    CHK_CU(hipFree(dThreadBlockKernelTransits[deviceIdx]));
    CHK_CU(hipFree(dSubWarpKernelTransits[deviceIdx]));
  }
  
  #if 0
  for (int s = 1; s < App().steps() - 2; s++) {
    std::unordered_set<VertexID_t> s1, s2, intersection;
    for (int i = 100000; i < 200000; i++) {
      VertexID_t v1 = hAllSamplesToTransitMapKeys[s+1][i];
      VertexID_t v2 = hAllTransitToSampleMapValues[s+2][i];
      //printf("v1 %d v2 %d\n", v1, v2);
      s1.insert(v1);
      s2.insert(v2);
    }
    
    for (auto e : s1) {
      if (s2.count(e) == 1) intersection.insert(e);
    }

    std::cout << "s: " << s << " intersection: " << intersection.size() << std::endl;
  }
  #endif
  return true;
}


template<class SampleType, typename App>
bool doSampleParallelSampling(CSR* csr, NextDoorData<SampleType, App>& nextDoorData)
{
  //Size of each sample output
  int finalSampleSize = getFinalSampleSize<App>();
  int neighborsToSampleAtStep = App().initialSampleSize(csr);
  size_t numDevices = nextDoorData.devices.size();

  std::vector<EdgePos_t*> hSumNeighborhoodSizes = std::vector<EdgePos_t*>(nextDoorData.devices.size());
  std::vector<EdgePos_t*> dSumNeighborhoodSizes = std::vector<EdgePos_t*>(nextDoorData.devices.size());
  std::vector<EdgePos_t*> dSampleNeighborhoodPos = std::vector<EdgePos_t*>(nextDoorData.devices.size());
  std::vector<VertexID_t*> dCollectiveNeighborhoodCSRCols = std::vector<VertexID_t*>(nextDoorData.devices.size());
  std::vector<EdgePos_t*> dCollectiveNeighborhoodCSRRows = std::vector<EdgePos_t*>(nextDoorData.devices.size());

  if (App().samplingType() == SamplingType::CollectiveNeighborhood) {
    for(auto deviceIdx = 0; deviceIdx < nextDoorData.devices.size(); deviceIdx++) {
    auto device = nextDoorData.devices[deviceIdx];
      CHK_CU(hipSetDevice(device));
      const size_t perDeviceNumSamples = PartDivisionSize(nextDoorData.samples.size(), deviceIdx, numDevices);
      CHK_CU(hipHostMalloc(&hSumNeighborhoodSizes[deviceIdx], sizeof(EdgePos_t)));
      CHK_CU(hipMalloc(&dSumNeighborhoodSizes[deviceIdx], sizeof(EdgePos_t)));
      CHK_CU(hipMalloc(&dSampleNeighborhoodPos[deviceIdx], sizeof(EdgePos_t)*perDeviceNumSamples));
    }
  }
  std::vector<GPUCSRPartition>& gpuCSRPartitions = nextDoorData.gpuCSRPartitions;
  std::vector<size_t> totalThreads = std::vector<size_t>(nextDoorData.devices.size());
  
  double end_to_end_t1 = convertTimeValToDouble(getTimeOfDay ());
  double collectiveNeighborhoodTime = 0.0f;

  for (int step = 0; step < App().steps(); step++) {
    //Number of threads created are equal to number of new neighbors to be sampled at a step.
    //In collective neighborhood we sample stepSize(step) vertices at each step
    //Otherwise need to sample product.

    const size_t numTransits = (App().samplingType() == CollectiveNeighborhood) ? 1 : neighborsToSampleAtStep;
    neighborsToSampleAtStep = (App().samplingType() == CollectiveNeighborhood) ? App().stepSize(step) : neighborsToSampleAtStep * App().stepSize(step);
    // std::cout << "totalThreads " << totalThreads << std::endl;
    double collectiveNeighborhood_t0 = convertTimeValToDouble(getTimeOfDay());
    
    for(auto deviceIdx = 0; deviceIdx < nextDoorData.devices.size(); deviceIdx++) {
      auto device = nextDoorData.devices[deviceIdx];
      CHK_CU(hipSetDevice(device));
      const size_t perDeviceNumSamples = PartDivisionSize(nextDoorData.samples.size(), deviceIdx, numDevices);
      const size_t deviceSampleStartPtr = PartStartPointer(nextDoorData.samples.size(), deviceIdx, numDevices);
      totalThreads[deviceIdx] = perDeviceNumSamples*neighborsToSampleAtStep;
      if (App().samplingType() == SamplingType::CollectiveNeighborhood) {
        assert(nextDoorData.devices.size() == 1);
        //FIXME: CollectiveNeighborhood Sampling for more than one GPU in Sample Parallel is not support 
        //Create collective neighborhood for all transits related to a sample
        collectiveNeighbrsSize<App><<<nextDoorData.samples.size(), N_THREADS>>>(step, nextDoorData.gpuCSRPartitions[0], 
                                                                            nextDoorData.INVALID_VERTEX,
                                                                            nextDoorData.dInitialSamples[deviceIdx], 
                                                                            nextDoorData.dFinalSamples[deviceIdx], 
                                                                            nextDoorData.samples.size(),
                                                                            dSampleNeighborhoodPos[deviceIdx],
                                                                            dSumNeighborhoodSizes[deviceIdx]);
        CHK_CU(hipGetLastError());
        CHK_CU(hipDeviceSynchronize());
        double __t1 = convertTimeValToDouble(getTimeOfDay());
        //TODO: Neighborhood is edges of all transit vertices. Hence, neighborhood size is (# of Transit Vertices)/(|G.V|) * |G.E|
        CHK_CU(hipMemcpy(hSumNeighborhoodSizes[deviceIdx], dSumNeighborhoodSizes[deviceIdx], sizeof(EdgePos_t), hipMemcpyDeviceToHost));
        CHK_CU(hipMalloc(&dCollectiveNeighborhoodCSRCols[deviceIdx], sizeof(VertexID_t)*(*hSumNeighborhoodSizes[deviceIdx])));
        CHK_CU(hipMalloc(&dCollectiveNeighborhoodCSRRows[deviceIdx], sizeof(EdgePos_t)*App().initialSampleSize(csr)*nextDoorData.samples.size()));
        double __t2 = convertTimeValToDouble(getTimeOfDay());
        
        collectiveNeighborhood<App><<<nextDoorData.samples.size(), N_THREADS>>>(step, nextDoorData.gpuCSRPartitions[0], 
                                                                            nextDoorData.INVALID_VERTEX,
                                                                            nextDoorData.dInitialSamples[deviceIdx],
                                                                            nextDoorData.dFinalSamples[deviceIdx], 
                                                                            nextDoorData.samples.size(),
                                                                            dCollectiveNeighborhoodCSRRows[deviceIdx],
                                                                            dCollectiveNeighborhoodCSRCols[deviceIdx],
                                                                            dSampleNeighborhoodPos[deviceIdx],
                                                                            dSumNeighborhoodSizes[deviceIdx]);
        CHK_CU(hipGetLastError());
        CHK_CU(hipDeviceSynchronize());
        
    #if 0
        //Check if the CSR is correct
        EdgePos_t* csrRows = new EdgePos_t[App().initialSampleSize(csr)*nextDoorData.samples.size()];
        EdgePos_t* csrCols = new VertexID_t[(*hSumNeighborhoodSizes)];
        EdgePos_t* samplePos = new EdgePos_t[nextDoorData.samples.size()];
        
        CHK_CU(hipMemcpy(csrCols, dCollectiveNeighborhoodCSRCols, sizeof(VertexID_t)*(*hSumNeighborhoodSizes), 
                          hipMemcpyDeviceToHost));
        CHK_CU(hipMemcpy(csrRows, dCollectiveNeighborhoodCSRRows, sizeof(EdgePos_t)*App().initialSampleSize(csr)*nextDoorData.samples.size(), 
                          hipMemcpyDeviceToHost));
        CHK_CU(hipMemcpy(samplePos, dSampleNeighborhoodPos, sizeof(EdgePos_t)*nextDoorData.samples.size(), 
                          hipMemcpyDeviceToHost));
        const int SZ = App().initialSampleSize(csr)*nextDoorData.samples.size();
        for (int sample = 0; sample < nextDoorData.samples.size(); sample++) {
          for (int v = 0; v < App().initialSampleSize(csr); v++) {
            EdgePos_t edgeStart = csrRows[sample * App().initialSampleSize(csr) + v];
            EdgePos_t edgeEnd = -1;
            EdgePos_t idxInRows = sample * App().initialSampleSize(csr) + v;
            
            //TODO: Add one more field to a vertex to each sample that is the length of all edges.
            if (v + 1 == App().initialSampleSize(csr)) {
              continue;
            }
            if (v + 1 < App().initialSampleSize(csr)) {
              edgeEnd = csrRows[idxInRows + 1];
            } else if (sample + 1 < nextDoorData.samples.size()) {
              edgeEnd = samplePos[sample + 1];
            } else {
              edgeEnd = (*hSumNeighborhoodSizes);
            }
              
            VertexID transit = nextDoorData.initialContents[sample * App().initialSampleSize(csr) + v];
            if (edgeEnd - edgeStart != csr->n_edges_for_vertex(transit)) {
              printf("transit %d edgeEnd %d edgeStart %d csr->n_edges_for_vertex(transit) %d\n", transit, edgeEnd, edgeStart, csr->n_edges_for_vertex(transit));
            }
            assert(edgeEnd - edgeStart == csr->n_edges_for_vertex(transit));
          }
        }
    #endif
        /*Sorting takes a ton of time (2-3x more). So, it probably be benificial to 
          * create a CSR matrix of the neighborhood of transit vertices.*/
        //Sort these edges of neighborhood
        /****************************
        void* dTempStorage = nullptr;
        size_t dTempStorageBytes = 0;
        hipcub::DeviceSegmentedRadixSort::SortKeys(dTempStorage, dTempStorageBytes, (const VertexID_t*)dCollectiveNeighborhood, 
                                                dCollectiveNeighborhood + sizeof(VertexID_t)*(*hSumNeighborhoodSizes), 
                                                *hSumNeighborhoodSizes, (int)nextDoorData.samples.size(),
                                                dSampleNeighborhoodPos, dSampleNeighborhoodPos + 1, 0, nextDoorData.maxBits);
        
        CHK_CU(hipMalloc(&dTempStorage, dTempStorageBytes));
        hipcub::DeviceSegmentedRadixSort::SortKeys(dTempStorage, dTempStorageBytes, (const VertexID_t*)dCollectiveNeighborhood, 
                                                dCollectiveNeighborhood + sizeof(VertexID_t)*(*hSumNeighborhoodSizes), 
                                                *hSumNeighborhoodSizes, (int)nextDoorData.samples.size(),
                                                dSampleNeighborhoodPos, dSampleNeighborhoodPos + 1, 0, nextDoorData.maxBits);
        CHK_CU(hipGetLastError());
        CHK_CU(hipDeviceSynchronize());
        ****************************/
      }

      double collectiveNeighborhood_t1 = convertTimeValToDouble(getTimeOfDay());
      collectiveNeighborhoodTime += (collectiveNeighborhood_t1 - collectiveNeighborhood_t0);

      if (App().hasExplicitTransits() and step > 0) {
        const size_t totalThreads = App().numSamples(csr)*numTransits;
        for (int _thExecs = 0; _thExecs < totalThreads; _thExecs += nextDoorData.maxThreadsPerKernel[deviceIdx]) {
          const size_t currExecThreads = min(nextDoorData.maxThreadsPerKernel[deviceIdx], totalThreads - _thExecs);

          explicitTransitsKernel<SampleType, App, false><<<DIVUP(currExecThreads, N_THREADS), N_THREADS>>>(step, gpuCSRPartitions[deviceIdx], 
                                                                                                      nextDoorData.INVALID_VERTEX,
                                                                                                      _thExecs, currExecThreads,
                                                                                                      totalThreads,
                                                                                                      nextDoorData.dOutputSamples[deviceIdx],
                                                                                                      nextDoorData.samples.size(),
                                                                                                      nullptr,
                                                                                                      nextDoorData.dSamplesToTransitMapValues[deviceIdx],
                                                                                                      nextDoorData.dCurandStates[deviceIdx]);
          
          CHK_CU(hipGetLastError());
          CHK_CU(hipDeviceSynchronize());
        }
      }

      //Perform SampleParallel Sampling
      sampleParallelKernel<SampleType, App, 256, false><<<min(1024L, nextDoorData.maxThreadsPerKernel[deviceIdx]/1024L), 256L>>>(step, gpuCSRPartitions[deviceIdx], 
                    deviceSampleStartPtr, nextDoorData.INVALID_VERTEX, totalThreads[deviceIdx], 
                    nextDoorData.dInitialSamples[deviceIdx], nextDoorData.dOutputSamples[deviceIdx], perDeviceNumSamples,
                    nextDoorData.dFinalSamples[deviceIdx], finalSampleSize, 
                    nextDoorData.dSamplesToTransitMapKeys[deviceIdx],
                    nextDoorData.dSamplesToTransitMapValues[deviceIdx],
                    nextDoorData.dSampleInsertionPositions[deviceIdx], nextDoorData.dCurandStates[deviceIdx]);
      CHK_CU(hipGetLastError());
      CHK_CU(hipDeviceSynchronize());
    }

    if (App().samplingType() == SamplingType::CollectiveNeighborhood) {
      for(auto deviceIdx = 0; deviceIdx < nextDoorData.devices.size(); deviceIdx++) {
        auto device = nextDoorData.devices[deviceIdx];
        CHK_CU(hipSetDevice(device));
        CHK_CU(hipFree(dCollectiveNeighborhoodCSRCols[deviceIdx]));
        CHK_CU(hipFree(dCollectiveNeighborhoodCSRRows[deviceIdx]));
      }
    }
  }

  double end_to_end_t2 = convertTimeValToDouble(getTimeOfDay ());
  
  std::cout << "SampleParallel: End to end time " << (end_to_end_t2 - end_to_end_t1) << " secs" << std::endl;
  if (App().samplingType() == SamplingType::CollectiveNeighborhood) {
    std::cout << "Collective Neighborhood Computing " << collectiveNeighborhoodTime << " secs" << std::endl;
  }

  if (App().samplingType() == SamplingType::CollectiveNeighborhood) {
    for(auto deviceIdx = 0; deviceIdx < nextDoorData.devices.size(); deviceIdx++) {
      auto device = nextDoorData.devices[deviceIdx];
      CHK_CU(hipHostFree(hSumNeighborhoodSizes[deviceIdx]));
      CHK_CU(hipFree(dSumNeighborhoodSizes[deviceIdx]));
      CHK_CU(hipFree(dSampleNeighborhoodPos[deviceIdx]));
    }
  }

  return true;
}

template<class SampleType, typename App>
std::vector<VertexID_t>& getFinalSamples(NextDoorData<SampleType, App>& nextDoorData)
{
  for(auto deviceIdx = 0; deviceIdx < nextDoorData.devices.size(); deviceIdx++) {
    auto device = nextDoorData.devices[deviceIdx];
    //Per Device Allocation
    CHK_CU(hipSetDevice(device));
    const size_t numSamples = nextDoorData.samples.size();
    const size_t finalSampleSize = getFinalSampleSize<App>();

    const size_t perDeviceNumSamples = PartDivisionSize(numSamples, deviceIdx, nextDoorData.devices.size());
    const size_t deviceSampleStartPtr = PartStartPointer(numSamples, deviceIdx, nextDoorData.devices.size());

    CHK_CU(hipMemcpy(&nextDoorData.hFinalSamples[0] + finalSampleSize * deviceSampleStartPtr, nextDoorData.dFinalSamples[deviceIdx], 
                      sizeof(nextDoorData.hFinalSamples[0]) * finalSampleSize * perDeviceNumSamples, hipMemcpyDeviceToHost));
    CHK_CU(hipMemcpy(&nextDoorData.samples[0] + deviceSampleStartPtr, nextDoorData.dOutputSamples[deviceIdx], 
                      perDeviceNumSamples*sizeof(SampleType), hipMemcpyDeviceToHost));
    // int i = 0;
    // printf("CHecking for invalidvertex %d\n", nextDoorData.INVALID_VERTEX);
    // for (auto v : nextDoorData.hFinalSamples) {
    //   if (v==nextDoorData.INVALID_VERTEX) {printf("i %d\n", i);break;}
    // i++;
    // }
  }

  return nextDoorData.hFinalSamples;
}

template<class SampleType, typename App>
bool nextdoor(const char* graph_file, const char* graph_type, const char* graph_format, 
             const int nruns, const bool chk_results, const bool print_samples,
             const char* kernelType, const bool enableLoadBalancing,
             bool (*checkResultsFunc)(NextDoorData<SampleType, App>&))
{
  std::vector<Vertex> vertices;

  //Load Graph
  Graph graph;
  CSR* csr;
  if ((csr = loadGraph(graph, (char*)graph_file, (char*)graph_type, (char*)graph_format)) == nullptr) {
    return false;
  }

  std::cout << "Graph has " <<graph.get_n_edges () << " edges and " << 
      graph.get_vertices ().size () << " vertices " << std::endl; 

  //graph.print(std::cout);
  NextDoorData<SampleType, App> nextDoorData;

  nextDoorData.csr = csr;
  allocNextDoorDataOnGPU<SampleType, App>(csr, nextDoorData);
  std::vector<GPUCSRPartition> gpuCSRPartitions = transferCSRToGPUs(nextDoorData, csr);
  nextDoorData.gpuCSRPartitions = gpuCSRPartitions;

  for (int i = 0; i < nruns; i++) {
    if (strcmp(kernelType, "TransitParallel") == 0)
      doTransitParallelSampling<SampleType, App>(csr, nextDoorData, enableLoadBalancing);
    else if (strcmp(kernelType, "SampleParallel") == 0)
      doSampleParallelSampling<SampleType, App>(csr, nextDoorData);
    else
      abort();
  }
    

  getFinalSamples(nextDoorData);

  size_t maxNeighborsToSample = 1;
  for (int step = 0; step < App().steps(); step++) {
    maxNeighborsToSample *= App().stepSize(step);
  }

  size_t finalSampleSize = getFinalSampleSize<App>();
  
  size_t totalSampledVertices = 0;

  for (auto s : nextDoorData.hFinalSamples) {
    totalSampledVertices += (int)(s != nextDoorData.INVALID_VERTEX);
  }

  if (print_samples) {
    for (size_t s = 0; s < nextDoorData.hFinalSamples.size(); s += finalSampleSize) {
      std::cout << "Contents of sample " << s/finalSampleSize << " [";
      for(size_t v = s; v < s + finalSampleSize; v++)
        std::cout << nextDoorData.hFinalSamples[v] << ", ";
      std::cout << "]" << std::endl;
    }
  }

  std::cout << "totalSampledVertices " << totalSampledVertices << std::endl;
  freeDeviceData(nextDoorData);
  if (chk_results) {
      return checkResultsFunc(nextDoorData);
  }
  
  return true;
}

#endif

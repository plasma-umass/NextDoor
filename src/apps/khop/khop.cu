struct KHopApp {

  __host__ __device__ int steps() {return 2;}

  __host__ __device__ 
  int stepSize(int k) {
    return ((k == 0) ? 25 : 10);
  }

  template<typename SampleType, typename EdgeArray, typename WeightArray>
  __device__ inline
  VertexID next(int step, CSRPartition* csr, const VertexID* transit, const VertexID sampleIdx,
                SampleType* sample, 
                const float max_weight,
                EdgeArray& transitEdges, WeightArray& transitEdgeWeights,
                const EdgePos_t numEdges, const VertexID_t neighbrID, hiprandState* state)
  {
    if (numEdges == 0) {
      return -1;
    }
    EdgePos_t id = RandNumGen::rand_int(state, numEdges);
    return transitEdges[id];
  }

  __host__ __device__ int samplingType()
  {
    return SamplingType::IndividualNeighborhood;
  }

  __host__ __device__ OutputFormat outputFormat()
  {
    return SampledVertices;
  }

  #define VERTICES_PER_SAMPLE 1

  __host__ __device__ EdgePos_t numSamples(CSR* graph)
  {
    return graph->get_n_vertices() / VERTICES_PER_SAMPLE;
  }

  template<class SampleType>
  __host__ std::vector<VertexID_t> initialSample(int sampleIdx, CSR* graph, SampleType& sample)
  {
    std::vector<VertexID_t> initialValue;

    for (int i = 0; i < VERTICES_PER_SAMPLE; i++) {
      initialValue.push_back(sampleIdx);//(rand())%graph->get_n_vertices());
    }

    return initialValue;
  }

  __host__ __device__ EdgePos_t initialSampleSize(CSR* graph)
  {
    return VERTICES_PER_SAMPLE;
  }

  __host__ __device__ bool hasExplicitTransits()
  {
    return false;
  }

  template<class SampleType>
  __host__ __device__ VertexID_t stepTransits(int step, const VertexID_t sampleID, SampleType& sample, int transitIdx, hiprandState* randState)
  {
    return -1;
  }

  template<class SampleType>
  __host__ SampleType initializeSample(CSR* graph, const VertexID_t sampleID)
  {
    SampleType sample;

    return sample;
  }
};

class KHopSample
{

};